#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/util/Log.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

namespace en
{
    VkDescriptorSetLayout NrcHpmRenderer::m_DescSetLayout;
    VkDescriptorPool NrcHpmRenderer::m_DescPool;

    PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR = nullptr;

    HANDLE GetSemaphoreHandle(VkDevice device, VkSemaphore vkSemaphore)
    {
        if (fpGetSemaphoreWin32HandleKHR == nullptr)
        {
            fpGetSemaphoreWin32HandleKHR = (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(device, "vkGetSemaphoreWin32HandleKHR");
        }

        VkSemaphoreGetWin32HandleInfoKHR vulkanSemaphoreGetWin32HandleInfoKHR = {};
        vulkanSemaphoreGetWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
        vulkanSemaphoreGetWin32HandleInfoKHR.pNext = NULL;
        vulkanSemaphoreGetWin32HandleInfoKHR.semaphore = vkSemaphore;
        vulkanSemaphoreGetWin32HandleInfoKHR.handleType = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;

        HANDLE handle;
        fpGetSemaphoreWin32HandleKHR(device, &vulkanSemaphoreGetWin32HandleInfoKHR, &handle);
        return handle;
    }

    void NrcHpmRenderer::Init(VkDevice device)
    {
        // Create desc set layout
        VkDescriptorSetLayoutBinding outputImageBinding;
        outputImageBinding.binding = 0;
        outputImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        outputImageBinding.descriptorCount = 1;
        outputImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        outputImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding primaryRayColorImageBinding;
        primaryRayColorImageBinding.binding = 1;
        primaryRayColorImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayColorImageBinding.descriptorCount = 1;
        primaryRayColorImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        primaryRayColorImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding primaryRayInfoImageBinding;
        primaryRayInfoImageBinding.binding = 2;
        primaryRayInfoImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayInfoImageBinding.descriptorCount = 1;
        primaryRayInfoImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        primaryRayInfoImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcRayOriginImageBinding;
        nrcRayOriginImageBinding.binding = 3;
        nrcRayOriginImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayOriginImageBinding.descriptorCount = 1;
        nrcRayOriginImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcRayOriginImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcRayDirImageBinding;
        nrcRayDirImageBinding.binding = 4;
        nrcRayDirImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayDirImageBinding.descriptorCount = 1;
        nrcRayDirImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcRayDirImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcInferInputBufferBinding;
        nrcInferInputBufferBinding.binding = 5;
        nrcInferInputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferInputBufferBinding.descriptorCount = 1;
        nrcInferInputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcInferInputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcInferOutputBufferBinding;
        nrcInferOutputBufferBinding.binding = 6;
        nrcInferOutputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferOutputBufferBinding.descriptorCount = 1;
        nrcInferOutputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcInferOutputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcTrainInputBufferBinding;
        nrcTrainInputBufferBinding.binding = 7;
        nrcTrainInputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainInputBufferBinding.descriptorCount = 1;
        nrcTrainInputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcTrainInputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcTrainTargetBufferBinding;
        nrcTrainTargetBufferBinding.binding = 8;
        nrcTrainTargetBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainTargetBufferBinding.descriptorCount = 1;
        nrcTrainTargetBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcTrainTargetBufferBinding.pImmutableSamplers = nullptr;

        std::vector<VkDescriptorSetLayoutBinding> bindings = {
                outputImageBinding,
                primaryRayColorImageBinding,
                primaryRayInfoImageBinding,
                nrcRayOriginImageBinding,
                nrcRayDirImageBinding,
                nrcInferInputBufferBinding,
                nrcInferOutputBufferBinding,
                nrcTrainInputBufferBinding,
                nrcTrainTargetBufferBinding };

        VkDescriptorSetLayoutCreateInfo layoutCI;
        layoutCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
        layoutCI.pNext = nullptr;
        layoutCI.flags = 0;
        layoutCI.bindingCount = bindings.size();
        layoutCI.pBindings = bindings.data();

        VkResult result = vkCreateDescriptorSetLayout(device, &layoutCI, nullptr, &m_DescSetLayout);
        ASSERT_VULKAN(result);

        // Create desc pool
        VkDescriptorPoolSize storageImagePS;
        storageImagePS.type = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        storageImagePS.descriptorCount = 5;

        VkDescriptorPoolSize storageBufferPS;
        storageBufferPS.type = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        storageBufferPS.descriptorCount = 4;

        std::vector<VkDescriptorPoolSize> poolSizes = { storageImagePS, storageBufferPS };

        VkDescriptorPoolCreateInfo poolCI;
        poolCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_POOL_CREATE_INFO;
        poolCI.pNext = nullptr;
        poolCI.flags = 0;
        poolCI.maxSets = 1;
        poolCI.poolSizeCount = poolSizes.size();
        poolCI.pPoolSizes = poolSizes.data();

        result = vkCreateDescriptorPool(device, &poolCI, nullptr, &m_DescPool);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::Shutdown(VkDevice device)
    {
        vkDestroyDescriptorPool(device, m_DescPool, nullptr);
        vkDestroyDescriptorSetLayout(device, m_DescSetLayout, nullptr);
    }

    NrcHpmRenderer::NrcHpmRenderer(
            uint32_t width,
            uint32_t height,
            uint32_t trainWidth,
            uint32_t trainHeight,
            const Camera& camera,
            const VolumeData& volumeData,
            const DirLight& dirLight,
            const PointLight& pointLight,
            const HdrEnvMap& hdrEnvMap,
            NeuralRadianceCache& nrc)
            :
            m_RenderWidth(width),
            m_RenderHeight(height),
            m_TrainWidth(trainWidth),
            m_TrainHeight(trainHeight),
            m_GenRaysShader("nrc/gen_rays.comp", false),
            m_PrepRayInfoShader("nrc/prep_ray_info.comp", false),
            m_PrepTrainRaysShader("nrc/prep_train_rays.comp", false),
            m_RenderShader("nrc/render.comp", false),
            m_CommandPool(VK_COMMAND_POOL_CREATE_RESET_COMMAND_BUFFER_BIT, VulkanAPI::GetGraphicsQFI()),
            m_Camera(camera),
            m_VolumeData(volumeData),
            m_DirLight(dirLight),
            m_PointLight(pointLight),
            m_HdrEnvMap(hdrEnvMap),
            m_Nrc(nrc)
    {
        VkDevice device = VulkanAPI::GetDevice();

        CreateSyncObjects(device);

        CreateNrcBuffers();

        m_Nrc.Init(
                m_RenderWidth * m_RenderHeight,
                m_TrainWidth * m_TrainHeight,
                reinterpret_cast<float*>(m_NrcInferInputDCuBuffer),
                reinterpret_cast<float*>(m_NrcInferOutputDCuBuffer),
                reinterpret_cast<float*>(m_NrcTrainInputDCuBuffer),
                reinterpret_cast<float*>(m_NrcTrainTargetDCuBuffer),
                m_CuExtCudaStartSemaphore,
                m_CuExtCudaFinishedSemaphore);

        m_CommandPool.AllocateBuffers(2, VK_COMMAND_BUFFER_LEVEL_PRIMARY);
        m_PreCudaCommandBuffer = m_CommandPool.GetBuffer(0);
        m_PostCudaCommandBuffer = m_CommandPool.GetBuffer(1);

        CreatePipelineLayout(device);

        InitSpecializationConstants();

        CreateGenRaysPipeline(device);
        CreatePrepRayInfoPipeline(device);
        CreatePrepTrainRaysPipeline(device);
        CreateRenderPipeline(device);

        CreateOutputImage(device);
        CreatePrimaryRayColorImage(device);
        CreatePrimaryRayInfoImage(device);
        CreateNrcRayOriginImage(device);
        CreateNrcRayDirImage(device);

        AllocateAndUpdateDescriptorSet(device);

        RecordPreCudaCommandBuffer();
        RecordPostCudaCommandBuffer();
    }

    void NrcHpmRenderer::Render(VkQueue queue) const
    {
        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 1;
        submitInfo.pSignalSemaphores = &m_CudaStartSemaphore;

        VkResult result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);

        m_Nrc.InferAndTrain();

        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 1;
        submitInfo.pWaitSemaphores = &m_CudaFinishedSemaphore;
        VkPipelineStageFlags waitStage = VK_PIPELINE_STAGE_ALL_COMMANDS_BIT;
        submitInfo.pWaitDstStageMask = &waitStage;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PostCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::Destroy()
    {
        VkDevice device = VulkanAPI::GetDevice();

        m_CommandPool.Destroy();

        vkDestroyImageView(device, m_NrcRayDirImageView, nullptr);
        vkFreeMemory(device, m_NrcRayDirImageMemory, nullptr);
        vkDestroyImage(device, m_NrcRayDirImage, nullptr);

        vkDestroyImageView(device, m_NrcRayOriginImageView, nullptr);
        vkFreeMemory(device, m_NrcRayOriginImageMemory, nullptr);
        vkDestroyImage(device, m_NrcRayOriginImage, nullptr);

        vkDestroyImageView(device, m_PrimaryRayInfoImageView, nullptr);
        vkFreeMemory(device, m_PrimaryRayInfoImageMemory, nullptr);
        vkDestroyImage(device, m_PrimaryRayInfoImage, nullptr);

        vkDestroyImageView(device, m_PrimaryRayColorImageView, nullptr);
        vkFreeMemory(device, m_PrimaryRayColorImageMemory, nullptr);
        vkDestroyImage(device, m_PrimaryRayColorImage, nullptr);

        vkDestroyImageView(device, m_OutputImageView, nullptr);
        vkFreeMemory(device, m_OutputImageMemory, nullptr);
        vkDestroyImage(device, m_OutputImage, nullptr);

        vkDestroyPipeline(device, m_RenderPipeline, nullptr);
        m_RenderShader.Destroy();

        vkDestroyPipeline(device, m_PrepTrainRaysPipeline, nullptr);
        m_PrepTrainRaysShader.Destroy();

        vkDestroyPipeline(device, m_PrepRayInfoPipeline, nullptr);
        m_PrepRayInfoShader.Destroy();

        vkDestroyPipeline(device, m_GenRaysPipeline, nullptr);
        m_GenRaysShader.Destroy();

        vkDestroyPipelineLayout(device, m_PipelineLayout, nullptr);

        m_NrcTrainTargetBuffer->Destroy();
        delete m_NrcTrainTargetBuffer;

        m_NrcTrainInputBuffer->Destroy();
        delete m_NrcTrainInputBuffer;

        m_NrcInferOutputBuffer->Destroy();
        delete m_NrcInferOutputBuffer;

        m_NrcInferInputBuffer->Destroy();
        delete m_NrcInferInputBuffer;

        vkDestroySemaphore(device, m_CudaFinishedSemaphore, nullptr);
        vkDestroySemaphore(device, m_CudaStartSemaphore, nullptr);
    }

    VkImage NrcHpmRenderer::GetImage() const
    {
        return m_OutputImage;
    }

    VkImageView NrcHpmRenderer::GetImageView() const
    {
        return m_OutputImageView;
    }

    void NrcHpmRenderer::CreateSyncObjects(VkDevice device)
    {
        // Create vk semaphore
        VkExportSemaphoreCreateInfoKHR vulkanExportSemaphoreCreateInfo = {};
        vulkanExportSemaphoreCreateInfo.sType = VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO_KHR;
        vulkanExportSemaphoreCreateInfo.pNext = nullptr;
        vulkanExportSemaphoreCreateInfo.handleTypes = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;

        VkSemaphoreCreateInfo semaphoreCI;
        semaphoreCI.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;
        semaphoreCI.pNext = &vulkanExportSemaphoreCreateInfo;
        semaphoreCI.flags = 0;

        VkResult result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &m_CudaStartSemaphore);
        ASSERT_VULKAN(result);

        result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &m_CudaFinishedSemaphore);
        ASSERT_VULKAN(result);

        // Export semaphore to cuda
        hipExternalSemaphoreHandleDesc extCudaSemaphoreHD{};
        extCudaSemaphoreHD.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

        extCudaSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, m_CudaStartSemaphore);
        hipError_t error = hipImportExternalSemaphore(&m_CuExtCudaStartSemaphore, &extCudaSemaphoreHD);
        ASSERT_CUDA(error);

        extCudaSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, m_CudaFinishedSemaphore);
        error = hipImportExternalSemaphore(&m_CuExtCudaFinishedSemaphore, &extCudaSemaphoreHD);
        ASSERT_CUDA(error);
    }

    void NrcHpmRenderer::CreateNrcBuffers()
    {
        // Calculate sizes
        m_NrcInferInputBufferSize = m_RenderWidth * m_RenderHeight * 5 * sizeof(float);
        m_NrcInferOutputBufferSize = m_RenderWidth * m_RenderHeight * 3 * sizeof(float);
        m_NrcTrainInputBufferSize = m_TrainWidth * m_TrainHeight * 5 * sizeof(float);
        m_NrcTrainTargetBufferSize = m_TrainWidth * m_TrainHeight * 3 * sizeof(float);

        // Create buffers
        m_NrcInferInputBuffer = new vk::Buffer(
                m_NrcInferInputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcInferOutputBuffer = new vk::Buffer(
                m_NrcInferOutputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcTrainInputBuffer = new vk::Buffer(
                m_NrcTrainInputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcTrainTargetBuffer = new vk::Buffer(
                m_NrcTrainTargetBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        // Get cuda external memory
        hipExternalMemoryHandleDesc cuExtMemHandleDesc{};
        cuExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;

        cuExtMemHandleDesc.handle.win32.handle = m_NrcInferInputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcInferInputBufferSize;
        hipError_t cudaResult = hipImportExternalMemory(&m_NrcInferInputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcInferOutputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcInferOutputBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcInferOutputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcTrainInputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcTrainInputBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcTrainInputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcTrainTargetBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcTrainTargetBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcTrainTargetCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        // Get cuda buffer
        hipExternalMemoryBufferDesc cudaExtBufferDesc{};
        cudaExtBufferDesc.offset = 0;
        cudaExtBufferDesc.flags = 0;

        cudaExtBufferDesc.size = m_NrcInferInputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcInferInputDCuBuffer, m_NrcInferInputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcInferOutputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcInferOutputDCuBuffer, m_NrcInferOutputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcTrainInputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcTrainInputDCuBuffer, m_NrcTrainInputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcTrainTargetBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcTrainTargetDCuBuffer, m_NrcTrainTargetCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);
    }

    void NrcHpmRenderer::CreatePipelineLayout(VkDevice device)
    {
        std::vector<VkDescriptorSetLayout> layouts = {
                Camera::GetDescriptorSetLayout(),
                VolumeData::GetDescriptorSetLayout(),
                DirLight::GetDescriptorSetLayout(),
                PointLight::GetDescriptorSetLayout(),
                HdrEnvMap::GetDescriptorSetLayout(),
                m_DescSetLayout };

        VkPipelineLayoutCreateInfo layoutCreateInfo;
        layoutCreateInfo.sType = VK_STRUCTURE_TYPE_PIPELINE_LAYOUT_CREATE_INFO;
        layoutCreateInfo.pNext = nullptr;
        layoutCreateInfo.flags = 0;
        layoutCreateInfo.setLayoutCount = layouts.size();
        layoutCreateInfo.pSetLayouts = layouts.data();
        layoutCreateInfo.pushConstantRangeCount = 0;
        layoutCreateInfo.pPushConstantRanges = nullptr;

        VkResult result = vkCreatePipelineLayout(device, &layoutCreateInfo, nullptr, &m_PipelineLayout);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::InitSpecializationConstants()
    {
        // Fill struct
        m_SpecData.renderWidth = m_RenderWidth;
        m_SpecData.renderHeight = m_RenderHeight;

        m_SpecData.trainWidth = m_TrainWidth;
        m_SpecData.trainHeight = m_TrainHeight;

        // Init map entries
        // Render size
        VkSpecializationMapEntry renderWidthEntry;
        renderWidthEntry.constantID = 0;
        renderWidthEntry.offset = offsetof(SpecializationData, SpecializationData::renderWidth);
        renderWidthEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry renderHeightEntry;
        renderHeightEntry.constantID = 1;
        renderHeightEntry.offset = offsetof(SpecializationData, SpecializationData::renderHeight);
        renderHeightEntry.size = sizeof(uint32_t);

        // Train size
        VkSpecializationMapEntry trainWidthEntry;
        trainWidthEntry.constantID = 2;
        trainWidthEntry.offset = offsetof(SpecializationData, SpecializationData::trainWidth);
        trainWidthEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry trainHeightEntry;
        trainHeightEntry.constantID = 3;
        trainHeightEntry.offset = offsetof(SpecializationData, SpecializationData::trainHeight);
        trainHeightEntry.size = sizeof(uint32_t);

        m_SpecMapEntries = {
                renderWidthEntry,
                renderHeightEntry,

                trainWidthEntry,
                trainHeightEntry };

        m_SpecInfo.mapEntryCount = m_SpecMapEntries.size();
        m_SpecInfo.pMapEntries = m_SpecMapEntries.data();
        m_SpecInfo.dataSize = sizeof(SpecializationData);
        m_SpecInfo.pData = &m_SpecData;
    }

    void NrcHpmRenderer::CreateGenRaysPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_GenRaysShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_GenRaysPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrepRayInfoPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_PrepRayInfoShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_PrepRayInfoPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrepTrainRaysPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_PrepTrainRaysShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_PrepTrainRaysPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateRenderPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_RenderShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_RenderPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateOutputImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_SAMPLED_BIT | VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_OutputImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_OutputImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_OutputImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_OutputImage, m_OutputImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_OutputImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_OutputImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_PreCudaCommandBuffer,
                m_OutputImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrimaryRayColorImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_PrimaryRayColorImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_PrimaryRayColorImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_PrimaryRayColorImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_PrimaryRayColorImage, m_PrimaryRayColorImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_PrimaryRayColorImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_PrimaryRayColorImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_PreCudaCommandBuffer,
                m_PrimaryRayColorImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrimaryRayInfoImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_PrimaryRayInfoImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_PrimaryRayInfoImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_PrimaryRayInfoImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_PrimaryRayInfoImage, m_PrimaryRayInfoImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_PrimaryRayInfoImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_PrimaryRayInfoImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_PreCudaCommandBuffer,
                m_PrimaryRayInfoImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateNrcRayOriginImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_NrcRayOriginImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_NrcRayOriginImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_NrcRayOriginImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_NrcRayOriginImage, m_NrcRayOriginImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_NrcRayOriginImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_NrcRayOriginImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_PreCudaCommandBuffer,
                m_NrcRayOriginImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateNrcRayDirImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_NrcRayDirImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_NrcRayDirImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_NrcRayDirImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_NrcRayDirImage, m_NrcRayDirImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_NrcRayDirImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_NrcRayDirImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_PreCudaCommandBuffer,
                m_NrcRayDirImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::AllocateAndUpdateDescriptorSet(VkDevice device)
    {
        // Allocate
        VkDescriptorSetAllocateInfo descSetAI;
        descSetAI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
        descSetAI.pNext = nullptr;
        descSetAI.descriptorPool = m_DescPool;
        descSetAI.descriptorSetCount = 1;
        descSetAI.pSetLayouts = &m_DescSetLayout;

        VkResult result = vkAllocateDescriptorSets(device, &descSetAI, &m_DescSet);
        ASSERT_VULKAN(result);

        // Write
        // Storage image writes
        VkDescriptorImageInfo outputImageInfo;
        outputImageInfo.sampler = VK_NULL_HANDLE;
        outputImageInfo.imageView = m_OutputImageView;
        outputImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet outputImageWrite;
        outputImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        outputImageWrite.pNext = nullptr;
        outputImageWrite.dstSet = m_DescSet;
        outputImageWrite.dstBinding = 0;
        outputImageWrite.dstArrayElement = 0;
        outputImageWrite.descriptorCount = 1;
        outputImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        outputImageWrite.pImageInfo = &outputImageInfo;
        outputImageWrite.pBufferInfo = nullptr;
        outputImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo primaryRayColorImageInfo;
        primaryRayColorImageInfo.sampler = VK_NULL_HANDLE;
        primaryRayColorImageInfo.imageView = m_PrimaryRayColorImageView;
        primaryRayColorImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet primaryRayColorImageWrite;
        primaryRayColorImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        primaryRayColorImageWrite.pNext = nullptr;
        primaryRayColorImageWrite.dstSet = m_DescSet;
        primaryRayColorImageWrite.dstBinding = 1;
        primaryRayColorImageWrite.dstArrayElement = 0;
        primaryRayColorImageWrite.descriptorCount = 1;
        primaryRayColorImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayColorImageWrite.pImageInfo = &primaryRayColorImageInfo;
        primaryRayColorImageWrite.pBufferInfo = nullptr;
        primaryRayColorImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo primaryRayInfoImageInfo;
        primaryRayInfoImageInfo.sampler = VK_NULL_HANDLE;
        primaryRayInfoImageInfo.imageView = m_PrimaryRayInfoImageView;
        primaryRayInfoImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet primaryRayInfoImageWrite;
        primaryRayInfoImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        primaryRayInfoImageWrite.pNext = nullptr;
        primaryRayInfoImageWrite.dstSet = m_DescSet;
        primaryRayInfoImageWrite.dstBinding = 2;
        primaryRayInfoImageWrite.dstArrayElement = 0;
        primaryRayInfoImageWrite.descriptorCount = 1;
        primaryRayInfoImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayInfoImageWrite.pImageInfo = &primaryRayInfoImageInfo;
        primaryRayInfoImageWrite.pBufferInfo = nullptr;
        primaryRayInfoImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo nrcRayOriginImageInfo;
        nrcRayOriginImageInfo.sampler = VK_NULL_HANDLE;
        nrcRayOriginImageInfo.imageView = m_NrcRayOriginImageView;
        nrcRayOriginImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet nrcRayOriginImageWrite;
        nrcRayOriginImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcRayOriginImageWrite.pNext = nullptr;
        nrcRayOriginImageWrite.dstSet = m_DescSet;
        nrcRayOriginImageWrite.dstBinding = 3;
        nrcRayOriginImageWrite.dstArrayElement = 0;
        nrcRayOriginImageWrite.descriptorCount = 1;
        nrcRayOriginImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayOriginImageWrite.pImageInfo = &nrcRayOriginImageInfo;
        nrcRayOriginImageWrite.pBufferInfo = nullptr;
        nrcRayOriginImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo nrcRayDirImageInfo;
        nrcRayDirImageInfo.sampler = VK_NULL_HANDLE;
        nrcRayDirImageInfo.imageView = m_NrcRayDirImageView;
        nrcRayDirImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet nrcRayDirImageWrite;
        nrcRayDirImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcRayDirImageWrite.pNext = nullptr;
        nrcRayDirImageWrite.dstSet = m_DescSet;
        nrcRayDirImageWrite.dstBinding = 4;
        nrcRayDirImageWrite.dstArrayElement = 0;
        nrcRayDirImageWrite.descriptorCount = 1;
        nrcRayDirImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayDirImageWrite.pImageInfo = &nrcRayDirImageInfo;
        nrcRayDirImageWrite.pBufferInfo = nullptr;
        nrcRayDirImageWrite.pTexelBufferView = nullptr;

        // Storage buffer writes
        VkDescriptorBufferInfo nrcInferInputBufferInfo;
        nrcInferInputBufferInfo.buffer = m_NrcInferInputBuffer->GetVulkanHandle();
        nrcInferInputBufferInfo.offset = 0;
        nrcInferInputBufferInfo.range = m_NrcInferInputBufferSize;

        VkWriteDescriptorSet nrcInferInputBufferWrite;
        nrcInferInputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcInferInputBufferWrite.pNext = nullptr;
        nrcInferInputBufferWrite.dstSet = m_DescSet;
        nrcInferInputBufferWrite.dstBinding = 5;
        nrcInferInputBufferWrite.dstArrayElement = 0;
        nrcInferInputBufferWrite.descriptorCount = 1;
        nrcInferInputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferInputBufferWrite.pImageInfo = nullptr;
        nrcInferInputBufferWrite.pBufferInfo = &nrcInferInputBufferInfo;
        nrcInferInputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcInferOutputBufferInfo;
        nrcInferOutputBufferInfo.buffer = m_NrcInferOutputBuffer->GetVulkanHandle();
        nrcInferOutputBufferInfo.offset = 0;
        nrcInferOutputBufferInfo.range = m_NrcInferOutputBufferSize;

        VkWriteDescriptorSet nrcInferOutputBufferWrite;
        nrcInferOutputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcInferOutputBufferWrite.pNext = nullptr;
        nrcInferOutputBufferWrite.dstSet = m_DescSet;
        nrcInferOutputBufferWrite.dstBinding = 6;
        nrcInferOutputBufferWrite.dstArrayElement = 0;
        nrcInferOutputBufferWrite.descriptorCount = 1;
        nrcInferOutputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferOutputBufferWrite.pImageInfo = nullptr;
        nrcInferOutputBufferWrite.pBufferInfo = &nrcInferOutputBufferInfo;
        nrcInferOutputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcTrainInputBufferInfo;
        nrcTrainInputBufferInfo.buffer = m_NrcTrainInputBuffer->GetVulkanHandle();
        nrcTrainInputBufferInfo.offset = 0;
        nrcTrainInputBufferInfo.range = m_NrcTrainInputBufferSize;

        VkWriteDescriptorSet nrcTrainInputBufferWrite;
        nrcTrainInputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcTrainInputBufferWrite.pNext = nullptr;
        nrcTrainInputBufferWrite.dstSet = m_DescSet;
        nrcTrainInputBufferWrite.dstBinding = 7;
        nrcTrainInputBufferWrite.dstArrayElement = 0;
        nrcTrainInputBufferWrite.descriptorCount = 1;
        nrcTrainInputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainInputBufferWrite.pImageInfo = nullptr;
        nrcTrainInputBufferWrite.pBufferInfo = &nrcTrainInputBufferInfo;
        nrcTrainInputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcTrainTargetBufferInfo;
        nrcTrainTargetBufferInfo.buffer = m_NrcTrainTargetBuffer->GetVulkanHandle();
        nrcTrainTargetBufferInfo.offset = 0;
        nrcTrainTargetBufferInfo.range = m_NrcTrainTargetBufferSize;

        VkWriteDescriptorSet nrcTrainTargetBufferWrite;
        nrcTrainTargetBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcTrainTargetBufferWrite.pNext = nullptr;
        nrcTrainTargetBufferWrite.dstSet = m_DescSet;
        nrcTrainTargetBufferWrite.dstBinding = 8;
        nrcTrainTargetBufferWrite.dstArrayElement = 0;
        nrcTrainTargetBufferWrite.descriptorCount = 1;
        nrcTrainTargetBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainTargetBufferWrite.pImageInfo = nullptr;
        nrcTrainTargetBufferWrite.pBufferInfo = &nrcTrainTargetBufferInfo;
        nrcTrainTargetBufferWrite.pTexelBufferView = nullptr;

        // Write writes
        std::vector<VkWriteDescriptorSet> writes = {
                outputImageWrite,
                primaryRayColorImageWrite,
                primaryRayInfoImageWrite,
                nrcRayOriginImageWrite,
                nrcRayDirImageWrite,
                nrcInferInputBufferWrite,
                nrcInferOutputBufferWrite,
                nrcTrainInputBufferWrite,
                nrcTrainTargetBufferWrite };

        vkUpdateDescriptorSets(device, writes.size(), writes.data(), 0, nullptr);
    }

    void NrcHpmRenderer::RecordPreCudaCommandBuffer()
    {
        // Begin
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        VkResult result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        // Collect descriptor sets
        std::vector<VkDescriptorSet> descSets = {
                m_Camera.GetDescriptorSet(),
                m_VolumeData.GetDescriptorSet(),
                m_DirLight.GetDescriptorSet(),
                m_PointLight.GetDescriptorSet(),
                m_HdrEnvMap.GetDescriptorSet(),
                m_DescSet };

        // Create memory barrier
        VkMemoryBarrier memoryBarrier;
        memoryBarrier.sType = VK_STRUCTURE_TYPE_MEMORY_BARRIER;
        memoryBarrier.pNext = nullptr;
        memoryBarrier.srcAccessMask = VK_ACCESS_MEMORY_WRITE_BIT;
        memoryBarrier.dstAccessMask = VK_ACCESS_MEMORY_READ_BIT;

        // Bind descriptor sets
        vkCmdBindDescriptorSets(
                m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PipelineLayout,
                0, descSets.size(), descSets.data(),
                0, nullptr);

        // Gen rays pipeline
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_GenRaysPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        vkCmdPipelineBarrier(
                m_PreCudaCommandBuffer,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_DEPENDENCY_DEVICE_GROUP_BIT,
                1, &memoryBarrier,
                0, nullptr,
                0, nullptr);

        // Prep ray info
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PrepRayInfoPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        vkCmdPipelineBarrier(
                m_PreCudaCommandBuffer,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_DEPENDENCY_DEVICE_GROUP_BIT,
                1, &memoryBarrier,
                0, nullptr,
                0, nullptr);

        // Prep train rays
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PrepTrainRaysPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_TrainWidth / 32, m_TrainHeight, 1);

        // End
        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::RecordPostCudaCommandBuffer()
    {
        // Begin command buffer
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        VkResult result = vkBeginCommandBuffer(m_PostCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        // Collect descriptor sets
        std::vector<VkDescriptorSet> descSets = {
                m_Camera.GetDescriptorSet(),
                m_VolumeData.GetDescriptorSet(),
                m_DirLight.GetDescriptorSet(),
                m_PointLight.GetDescriptorSet(),
                m_HdrEnvMap.GetDescriptorSet(),
                m_DescSet };

        // Create memory barrier
        VkMemoryBarrier memoryBarrier;
        memoryBarrier.sType = VK_STRUCTURE_TYPE_MEMORY_BARRIER;
        memoryBarrier.pNext = nullptr;
        memoryBarrier.srcAccessMask = VK_ACCESS_MEMORY_WRITE_BIT;
        memoryBarrier.dstAccessMask = VK_ACCESS_MEMORY_READ_BIT;

        // Bind descriptor sets
        vkCmdBindDescriptorSets(
                m_PostCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PipelineLayout,
                0, descSets.size(), descSets.data(),
                0, nullptr);

        // Render pipeline
        vkCmdBindPipeline(m_PostCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_RenderPipeline);
        vkCmdDispatch(m_PostCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        // End command buffer
        result = vkEndCommandBuffer(m_PostCudaCommandBuffer);
        ASSERT_VULKAN(result);
    }
}
