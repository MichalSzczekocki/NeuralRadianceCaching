#include "hip/hip_runtime.h"
#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/util/Log.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <glm/gtc/random.hpp>
#include <imgui.h>

#define TINYEXR_IMPLEMENTATION
#include <tinyexr.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

namespace en
{
    VkDescriptorSetLayout NrcHpmRenderer::m_DescSetLayout;
    VkDescriptorPool NrcHpmRenderer::m_DescPool;

    PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR = nullptr;

    HANDLE GetSemaphoreHandle(VkDevice device, VkSemaphore vkSemaphore)
    {
        if (fpGetSemaphoreWin32HandleKHR == nullptr)
        {
            fpGetSemaphoreWin32HandleKHR = (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(device, "vkGetSemaphoreWin32HandleKHR");
        }

        VkSemaphoreGetWin32HandleInfoKHR vulkanSemaphoreGetWin32HandleInfoKHR = {};
        vulkanSemaphoreGetWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
        vulkanSemaphoreGetWin32HandleInfoKHR.pNext = NULL;
        vulkanSemaphoreGetWin32HandleInfoKHR.semaphore = vkSemaphore;
        vulkanSemaphoreGetWin32HandleInfoKHR.handleType = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;

        HANDLE handle;
        fpGetSemaphoreWin32HandleKHR(device, &vulkanSemaphoreGetWin32HandleInfoKHR, &handle);
        return handle;
    }

    void NrcHpmRenderer::Init(VkDevice device)
    {
        // Create desc set layout
        uint32_t bindingIndex = 0;

        VkDescriptorSetLayoutBinding outputImageBinding;
        outputImageBinding.binding = bindingIndex++;
        outputImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        outputImageBinding.descriptorCount = 1;
        outputImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        outputImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding primaryRayColorImageBinding;
        primaryRayColorImageBinding.binding = bindingIndex++;
        primaryRayColorImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayColorImageBinding.descriptorCount = 1;
        primaryRayColorImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        primaryRayColorImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding primaryRayInfoImageBinding;
        primaryRayInfoImageBinding.binding = bindingIndex++;
        primaryRayInfoImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayInfoImageBinding.descriptorCount = 1;
        primaryRayInfoImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        primaryRayInfoImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcRayOriginImageBinding;
        nrcRayOriginImageBinding.binding = bindingIndex++;
        nrcRayOriginImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayOriginImageBinding.descriptorCount = 1;
        nrcRayOriginImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcRayOriginImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcRayDirImageBinding;
        nrcRayDirImageBinding.binding = bindingIndex++;
        nrcRayDirImageBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayDirImageBinding.descriptorCount = 1;
        nrcRayDirImageBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcRayDirImageBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcInferInputBufferBinding;
        nrcInferInputBufferBinding.binding = bindingIndex++;
        nrcInferInputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferInputBufferBinding.descriptorCount = 1;
        nrcInferInputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcInferInputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcInferOutputBufferBinding;
        nrcInferOutputBufferBinding.binding = bindingIndex++;
        nrcInferOutputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferOutputBufferBinding.descriptorCount = 1;
        nrcInferOutputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcInferOutputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcTrainInputBufferBinding;
        nrcTrainInputBufferBinding.binding = bindingIndex++;
        nrcTrainInputBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainInputBufferBinding.descriptorCount = 1;
        nrcTrainInputBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcTrainInputBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcTrainTargetBufferBinding;
        nrcTrainTargetBufferBinding.binding = bindingIndex++;
        nrcTrainTargetBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainTargetBufferBinding.descriptorCount = 1;
        nrcTrainTargetBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcTrainTargetBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcInferFilterBufferBinding;
        nrcInferFilterBufferBinding.binding = bindingIndex++;
        nrcInferFilterBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferFilterBufferBinding.descriptorCount = 1;
        nrcInferFilterBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcInferFilterBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding nrcTrainRayResBufferBinding;
        nrcTrainRayResBufferBinding.binding = bindingIndex++;
        nrcTrainRayResBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainRayResBufferBinding.descriptorCount = 1;
        nrcTrainRayResBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        nrcTrainRayResBufferBinding.pImmutableSamplers = nullptr;

        VkDescriptorSetLayoutBinding uniformBufferBinding;
        uniformBufferBinding.binding = bindingIndex++;
        uniformBufferBinding.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
        uniformBufferBinding.descriptorCount = 1;
        uniformBufferBinding.stageFlags = VK_SHADER_STAGE_COMPUTE_BIT;
        uniformBufferBinding.pImmutableSamplers = nullptr;

        std::vector<VkDescriptorSetLayoutBinding> bindings = {
                outputImageBinding,
                primaryRayColorImageBinding,
                primaryRayInfoImageBinding,
                nrcRayOriginImageBinding,
                nrcRayDirImageBinding,
                nrcInferInputBufferBinding,
                nrcInferOutputBufferBinding,
                nrcTrainInputBufferBinding,
                nrcTrainTargetBufferBinding,
                nrcInferFilterBufferBinding,
                nrcTrainRayResBufferBinding,
                uniformBufferBinding
        };

        VkDescriptorSetLayoutCreateInfo layoutCI;
        layoutCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
        layoutCI.pNext = nullptr;
        layoutCI.flags = 0;
        layoutCI.bindingCount = bindings.size();
        layoutCI.pBindings = bindings.data();

        VkResult result = vkCreateDescriptorSetLayout(device, &layoutCI, nullptr, &m_DescSetLayout);
        ASSERT_VULKAN(result);

        // Create desc pool
        VkDescriptorPoolSize storageImagePS;
        storageImagePS.type = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        storageImagePS.descriptorCount = 5;

        VkDescriptorPoolSize storageBufferPS;
        storageBufferPS.type = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        storageBufferPS.descriptorCount = 6;

        VkDescriptorPoolSize uniformBufferPS;
        uniformBufferPS.type = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
        uniformBufferPS.descriptorCount = 1;

        std::vector<VkDescriptorPoolSize> poolSizes = { storageImagePS, storageBufferPS, uniformBufferPS };

        VkDescriptorPoolCreateInfo poolCI;
        poolCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_POOL_CREATE_INFO;
        poolCI.pNext = nullptr;
        poolCI.flags = 0;
        poolCI.maxSets = 1;
        poolCI.poolSizeCount = poolSizes.size();
        poolCI.pPoolSizes = poolSizes.data();

        result = vkCreateDescriptorPool(device, &poolCI, nullptr, &m_DescPool);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::Shutdown(VkDevice device)
    {
        vkDestroyDescriptorPool(device, m_DescPool, nullptr);
        vkDestroyDescriptorSetLayout(device, m_DescSetLayout, nullptr);
    }

    NrcHpmRenderer::NrcHpmRenderer(
            uint32_t width,
            uint32_t height,
            float trainSampleRatio,
            uint32_t trainSpp,
            const Camera* camera,
            const HpmScene& hpmScene,
            NeuralRadianceCache& nrc)
            :
            m_RenderWidth(width),
            m_RenderHeight(height),
            m_TrainSpp(trainSpp),
            m_GenRaysShader("nrc/gen_rays.comp", false),
            m_PrepInferRaysShader("nrc/prep_infer_rays.comp", false),
            m_PrepTrainRaysShader("nrc/prep_train_rays.comp", false),
            m_RenderShader("nrc/render.comp", false),
            m_CommandPool(VK_COMMAND_POOL_CREATE_RESET_COMMAND_BUFFER_BIT, VulkanAPI::GetGraphicsQFI()),
            m_Camera(camera),
            m_HpmScene(hpmScene),
            m_Nrc(nrc),
            m_UniformBuffer(
                    sizeof(UniformData),
                    VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                    VK_BUFFER_USAGE_UNIFORM_BUFFER_BIT,
                    {})
    {
        // Calc train sample extent
        float sqrtTrainSampleRatio = std::sqrt(trainSampleRatio);
        m_TrainWidth = sqrtTrainSampleRatio * m_RenderWidth;
        m_TrainHeight = sqrtTrainSampleRatio * m_RenderHeight;
        m_TrainWidth -= m_TrainWidth % 16;
        m_TrainHeight -= m_TrainHeight % 16;

        // Init components
        VkDevice device = VulkanAPI::GetDevice();

        CreateSyncObjects(device);

        CreateNrcBuffers();

        m_Nrc.Init(
                m_RenderWidth * m_RenderHeight,
                m_TrainWidth * m_TrainHeight,
                reinterpret_cast<float*>(m_NrcInferInputDCuBuffer),
                reinterpret_cast<float*>(m_NrcInferOutputDCuBuffer),
                reinterpret_cast<float*>(m_NrcTrainInputDCuBuffer),
                reinterpret_cast<float*>(m_NrcTrainTargetDCuBuffer),
                m_CuExtCudaStartSemaphore,
                m_CuExtCudaFinishedSemaphore);

        m_NrcInferFilterBufferSize = sizeof(uint32_t) * m_Nrc.GetInferBatchCount();
        m_NrcInferFilterData = malloc(m_NrcInferFilterBufferSize);
        m_NrcInferFilterBuffer = new vk::Buffer(
                m_NrcInferFilterBufferSize,
                VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {});

        m_NrcTrainRayResBufferSize = 6 * sizeof(uint32_t) * m_TrainWidth * m_TrainHeight;
        m_NrcTrainRayResBuffer = new vk::Buffer(
                m_NrcTrainRayResBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT,
                {});

        m_CommandPool.AllocateBuffers(3, VK_COMMAND_BUFFER_LEVEL_PRIMARY);
        m_PreCudaCommandBuffer = m_CommandPool.GetBuffer(0);
        m_PostCudaCommandBuffer = m_CommandPool.GetBuffer(1);
        m_RandomTasksCmdBuf = m_CommandPool.GetBuffer(2);

        CreatePipelineLayout(device);

        InitSpecializationConstants();

        CreateGenRaysPipeline(device);
        CreatePrepInferRaysPipeline(device);
        CreatePrepTrainRaysPipeline(device);
        CreateRenderPipeline(device);

        CreateOutputImage(device);
        CreatePrimaryRayColorImage(device);
        CreatePrimaryRayInfoImage(device);
        CreateNrcRayOriginImage(device);
        CreateNrcRayDirImage(device);

        AllocateAndUpdateDescriptorSet(device);

        CreateQueryPool(device);

        RecordPreCudaCommandBuffer();
        RecordPostCudaCommandBuffer();
    }

    void NrcHpmRenderer::Render(VkQueue queue)
    {
        // Check if camera moved
        if (m_Camera->HasChanged()) { m_BlendIndex = 1; }

        // Calc blending factor
        m_UniformData.blendFactor = 1.0 / static_cast<float>(m_BlendIndex);

        // Generate random
        m_UniformData.random = glm::linearRand(glm::vec4(0.0f), glm::vec4(1.0f));

        // Update uniform buffer
        m_UniformBuffer.SetData(sizeof(UniformData), &m_UniformData, 0, 0);

        // Update blending index
        if (m_ShouldBlend) { m_BlendIndex++; }

        // Pre cuda
        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PreCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 1;
        submitInfo.pSignalSemaphores = &m_CudaStartSemaphore;

        VkResult result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);

        // Sync infer filter
        m_NrcInferFilterBuffer->GetData(m_NrcInferFilterBufferSize, m_NrcInferFilterData, 0, 0);

        // Cuda
        m_Nrc.InferAndTrain(reinterpret_cast<uint32_t*>(m_NrcInferFilterData));

        // Post cuda
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 1;
        submitInfo.pWaitSemaphores = &m_CudaFinishedSemaphore;
        VkPipelineStageFlags waitStage = VK_PIPELINE_STAGE_ALL_COMMANDS_BIT;
        submitInfo.pWaitDstStageMask = &waitStage;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_PostCudaCommandBuffer;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::Destroy()
    {
        VkDevice device = VulkanAPI::GetDevice();

        m_CommandPool.Destroy();

        m_UniformBuffer.Destroy();

        vkDestroyQueryPool(device, m_QueryPool, nullptr);

        vkDestroyImageView(device, m_NrcRayDirImageView, nullptr);
        vkFreeMemory(device, m_NrcRayDirImageMemory, nullptr);
        vkDestroyImage(device, m_NrcRayDirImage, nullptr);

        vkDestroyImageView(device, m_NrcRayOriginImageView, nullptr);
        vkFreeMemory(device, m_NrcRayOriginImageMemory, nullptr);
        vkDestroyImage(device, m_NrcRayOriginImage, nullptr);

        vkDestroyImageView(device, m_PrimaryRayInfoImageView, nullptr);
        vkFreeMemory(device, m_PrimaryRayInfoImageMemory, nullptr);
        vkDestroyImage(device, m_PrimaryRayInfoImage, nullptr);

        vkDestroyImageView(device, m_PrimaryRayColorImageView, nullptr);
        vkFreeMemory(device, m_PrimaryRayColorImageMemory, nullptr);
        vkDestroyImage(device, m_PrimaryRayColorImage, nullptr);

        vkDestroyImageView(device, m_OutputImageView, nullptr);
        vkFreeMemory(device, m_OutputImageMemory, nullptr);
        vkDestroyImage(device, m_OutputImage, nullptr);

        vkDestroyPipeline(device, m_RenderPipeline, nullptr);
        m_RenderShader.Destroy();

        vkDestroyPipeline(device, m_PrepTrainRaysPipeline, nullptr);
        m_PrepTrainRaysShader.Destroy();

        vkDestroyPipeline(device, m_PrepInferRaysPipeline, nullptr);
        m_PrepInferRaysShader.Destroy();

        vkDestroyPipeline(device, m_GenRaysPipeline, nullptr);
        m_GenRaysShader.Destroy();

        vkDestroyPipelineLayout(device, m_PipelineLayout, nullptr);

        m_NrcTrainRayResBuffer->Destroy();
        delete m_NrcTrainRayResBuffer;

        m_NrcInferFilterBuffer->Destroy();
        delete m_NrcInferFilterBuffer;
        delete m_NrcInferFilterData;

        m_NrcTrainTargetBuffer->Destroy();
        delete m_NrcTrainTargetBuffer;
        ASSERT_CUDA(hipDestroyExternalMemory(m_NrcTrainTargetCuExtMem));

        m_NrcTrainInputBuffer->Destroy();
        delete m_NrcTrainInputBuffer;
        ASSERT_CUDA(hipDestroyExternalMemory(m_NrcTrainInputCuExtMem));

        m_NrcInferOutputBuffer->Destroy();
        delete m_NrcInferOutputBuffer;
        ASSERT_CUDA(hipDestroyExternalMemory(m_NrcInferOutputCuExtMem));

        m_NrcInferInputBuffer->Destroy();
        delete m_NrcInferInputBuffer;
        ASSERT_CUDA(hipDestroyExternalMemory(m_NrcInferInputCuExtMem));

        vkDestroySemaphore(device, m_CudaFinishedSemaphore, nullptr);
        ASSERT_CUDA(hipDestroyExternalSemaphore(m_CuExtCudaFinishedSemaphore));

        vkDestroySemaphore(device, m_CudaStartSemaphore, nullptr);
        ASSERT_CUDA(hipDestroyExternalSemaphore(m_CuExtCudaStartSemaphore));
    }

    void NrcHpmRenderer::ExportOutputImageToFile(VkQueue queue, const std::string& filePath) const
    {
        const size_t floatCount = m_RenderWidth * m_RenderHeight * 4;
        const size_t bufferSize = floatCount * sizeof(float);

        vk::Buffer vkBuffer(
                bufferSize,
                VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {});

        VkCommandBufferBeginInfo cmdBufBI;
        cmdBufBI.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        cmdBufBI.pNext = nullptr;
        cmdBufBI.flags = 0;
        cmdBufBI.pInheritanceInfo = nullptr;
        ASSERT_VULKAN(vkBeginCommandBuffer(m_RandomTasksCmdBuf, &cmdBufBI));

        VkBufferImageCopy region;
        region.bufferOffset = 0;
        region.bufferRowLength = m_RenderWidth;
        region.bufferImageHeight = m_RenderHeight;
        region.imageSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        region.imageSubresource.mipLevel = 0;
        region.imageSubresource.baseArrayLayer = 0;
        region.imageSubresource.layerCount = 1;
        region.imageOffset = { 0, 0, 0 };
        region.imageExtent = { m_RenderWidth, m_RenderHeight, 1 };

        vkCmdCopyImageToBuffer(m_RandomTasksCmdBuf, m_OutputImage, VK_IMAGE_LAYOUT_GENERAL, vkBuffer.GetVulkanHandle(), 1, &region);

        ASSERT_VULKAN(vkEndCommandBuffer(m_RandomTasksCmdBuf));

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        ASSERT_VULKAN(vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE));
        ASSERT_VULKAN(vkQueueWaitIdle(queue));

        std::vector<float> buffer(floatCount);
        vkBuffer.GetData(bufferSize, buffer.data(), 0, 0);
        vkBuffer.Destroy();

        // Store in exr file
        if (TINYEXR_SUCCESS != SaveEXR(buffer.data(), m_RenderWidth, m_RenderHeight, 4, 0, filePath.c_str(), nullptr))
        {
            en::Log::Error("TINYEXR Error", true);
        }
    }

    void NrcHpmRenderer::EvaluateTimestampQueries()
    {
        VkDevice device = VulkanAPI::GetDevice();
        std::vector<uint64_t> queryResults(c_QueryCount);
        ASSERT_VULKAN(vkGetQueryPoolResults(
                device,
                m_QueryPool,
                0,
                c_QueryCount,
                sizeof(uint64_t) * c_QueryCount,
                queryResults.data(),
                sizeof(uint64_t),
                VK_QUERY_RESULT_64_BIT));
        vkResetQueryPool(device, m_QueryPool, 0, c_QueryCount);

        for (size_t i = 0; i < c_QueryCount - 1; i++)
        {
            m_TimePeriods[i] = c_TimestampPeriodInMS * static_cast<float>(queryResults[i + 1] - queryResults[i]);
        }
        m_TimePeriods[c_QueryCount - 1] = c_TimestampPeriodInMS * static_cast<float>(queryResults[c_QueryCount - 1] - queryResults[0]);
    }

    void NrcHpmRenderer::RenderImGui()
    {
        ImGui::Begin("NrcHpmRenderer");

        size_t periodIndex = 0;
        ImGui::Text("GenRays Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("PrepInferRays Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("PrepTrainRays Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("Cuda Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("Render Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("Total Time %f ms", m_TimePeriods[periodIndex++]);
        ImGui::Text("Theoretical FPS %f", 1000.0f / m_TimePeriods[c_QueryCount - 1]);

        ImGui::Checkbox("Show NRC", reinterpret_cast<bool*>(&m_UniformData.showNrc));

        ImGui::Checkbox("Blend", &m_ShouldBlend);
        ImGui::Text("Blend index %u", m_BlendIndex);
        if (ImGui::Button("Reset blending")) { m_BlendIndex = 1; }

        ImGui::End();
    }

    float NrcHpmRenderer::CompareReferenceMSE(VkQueue queue, const float* referenceData) const
    {
        const size_t floatCount = m_RenderWidth * m_RenderHeight * 4;
        const size_t bufferSize = floatCount * sizeof(float);

        vk::Buffer vkBuffer(
                bufferSize,
                VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {});

        VkCommandBufferBeginInfo cmdBufBI;
        cmdBufBI.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        cmdBufBI.pNext = nullptr;
        cmdBufBI.flags = 0;
        cmdBufBI.pInheritanceInfo = nullptr;
        ASSERT_VULKAN(vkBeginCommandBuffer(m_RandomTasksCmdBuf, &cmdBufBI));

        VkBufferImageCopy region;
        region.bufferOffset = 0;
        region.bufferRowLength = m_RenderWidth;
        region.bufferImageHeight = m_RenderHeight;
        region.imageSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        region.imageSubresource.mipLevel = 0;
        region.imageSubresource.baseArrayLayer = 0;
        region.imageSubresource.layerCount = 1;
        region.imageOffset = { 0, 0, 0 };
        region.imageExtent = { m_RenderWidth, m_RenderHeight, 1 };

        vkCmdCopyImageToBuffer(m_RandomTasksCmdBuf, m_OutputImage, VK_IMAGE_LAYOUT_GENERAL, vkBuffer.GetVulkanHandle(), 1, &region);

        ASSERT_VULKAN(vkEndCommandBuffer(m_RandomTasksCmdBuf));

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        ASSERT_VULKAN(vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE));
        ASSERT_VULKAN(vkQueueWaitIdle(queue));

        std::vector<float> buffer(floatCount);
        vkBuffer.GetData(bufferSize, buffer.data(), 0, 0);
        vkBuffer.Destroy();

        // Compare mse
        float mse = 0.0;
        for (size_t pixel = 0; pixel < m_RenderWidth * m_RenderHeight; pixel++)
        {
            const float referenceR = referenceData[(pixel * 4) + 0];
            const float referenceG = referenceData[(pixel * 4) + 1];
            const float referenceB = referenceData[(pixel * 4) + 2];
            const float referenceA = referenceData[(pixel * 4) + 3];

            const float rendererR = buffer[(pixel + 4) + 0];
            const float rendererG = buffer[(pixel + 4) + 1];
            const float rendererB = buffer[(pixel + 4) + 2];
            const float rendererA = buffer[(pixel + 4) + 3];

            if (referenceA == 1.0)
            {
                const float distanceR = referenceR - rendererR;
                const float distanceG = referenceG - rendererG;
                const float distanceB = referenceB - rendererB;
                mse += (distanceR * distanceR) + (distanceG * distanceG) + (distanceB * distanceB);
            }
        }
        mse /= 3.0f * m_RenderWidth * m_RenderHeight;

        return mse;
    }

    VkImage NrcHpmRenderer::GetImage() const
    {
        return m_OutputImage;
    }

    VkImageView NrcHpmRenderer::GetImageView() const
    {
        return m_OutputImageView;
    }

    void NrcHpmRenderer::SetCamera(const Camera* camera)
    {
        m_BlendIndex = 1;
        m_Camera = camera;
        RecordPreCudaCommandBuffer();
        RecordPostCudaCommandBuffer();
    }

    void NrcHpmRenderer::CreateSyncObjects(VkDevice device)
    {
        // Create vk semaphore
        VkExportSemaphoreCreateInfoKHR vulkanExportSemaphoreCreateInfo = {};
        vulkanExportSemaphoreCreateInfo.sType = VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO_KHR;
        vulkanExportSemaphoreCreateInfo.pNext = nullptr;
        vulkanExportSemaphoreCreateInfo.handleTypes = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;

        VkSemaphoreCreateInfo semaphoreCI;
        semaphoreCI.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;
        semaphoreCI.pNext = &vulkanExportSemaphoreCreateInfo;
        semaphoreCI.flags = 0;

        VkResult result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &m_CudaStartSemaphore);
        ASSERT_VULKAN(result);

        result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &m_CudaFinishedSemaphore);
        ASSERT_VULKAN(result);

        // Export semaphore to cuda
        hipExternalSemaphoreHandleDesc extCudaSemaphoreHD{};
        extCudaSemaphoreHD.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

        extCudaSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, m_CudaStartSemaphore);
        hipError_t error = hipImportExternalSemaphore(&m_CuExtCudaStartSemaphore, &extCudaSemaphoreHD);
        ASSERT_CUDA(error);

        extCudaSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, m_CudaFinishedSemaphore);
        error = hipImportExternalSemaphore(&m_CuExtCudaFinishedSemaphore, &extCudaSemaphoreHD);
        ASSERT_CUDA(error);
    }

    void NrcHpmRenderer::CreateNrcBuffers()
    {
        // Calculate sizes
        m_NrcInferInputBufferSize = m_RenderWidth * m_RenderHeight * 5 * sizeof(float);
        m_NrcInferOutputBufferSize = m_RenderWidth * m_RenderHeight * 3 * sizeof(float);
        m_NrcTrainInputBufferSize = m_TrainWidth * m_TrainHeight * 5 * sizeof(float);
        m_NrcTrainTargetBufferSize = m_TrainWidth * m_TrainHeight * 3 * sizeof(float);

        // Create buffers
        m_NrcInferInputBuffer = new vk::Buffer(
                m_NrcInferInputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcInferOutputBuffer = new vk::Buffer(
                m_NrcInferOutputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcTrainInputBuffer = new vk::Buffer(
                m_NrcTrainInputBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        m_NrcTrainTargetBuffer = new vk::Buffer(
                m_NrcTrainTargetBufferSize,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT,
                VK_BUFFER_USAGE_STORAGE_BUFFER_BIT | VK_BUFFER_USAGE_TRANSFER_DST_BIT,
                {},
                VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT);

        // Get cuda external memory
        hipExternalMemoryHandleDesc cuExtMemHandleDesc{};
        cuExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;

        cuExtMemHandleDesc.handle.win32.handle = m_NrcInferInputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcInferInputBufferSize;
        hipError_t cudaResult = hipImportExternalMemory(&m_NrcInferInputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcInferOutputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcInferOutputBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcInferOutputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcTrainInputBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcTrainInputBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcTrainInputCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        cuExtMemHandleDesc.handle.win32.handle = m_NrcTrainTargetBuffer->GetMemoryWin32Handle();
        cuExtMemHandleDesc.size = m_NrcTrainTargetBufferSize;
        cudaResult = hipImportExternalMemory(&m_NrcTrainTargetCuExtMem, &cuExtMemHandleDesc);
        ASSERT_CUDA(cudaResult);

        // Get cuda buffer
        hipExternalMemoryBufferDesc cudaExtBufferDesc{};
        cudaExtBufferDesc.offset = 0;
        cudaExtBufferDesc.flags = 0;

        cudaExtBufferDesc.size = m_NrcInferInputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcInferInputDCuBuffer, m_NrcInferInputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcInferOutputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcInferOutputDCuBuffer, m_NrcInferOutputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcTrainInputBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcTrainInputDCuBuffer, m_NrcTrainInputCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);

        cudaExtBufferDesc.size = m_NrcTrainTargetBufferSize;
        cudaResult = hipExternalMemoryGetMappedBuffer(&m_NrcTrainTargetDCuBuffer, m_NrcTrainTargetCuExtMem, &cudaExtBufferDesc);
        ASSERT_CUDA(cudaResult);
    }

    void NrcHpmRenderer::CreatePipelineLayout(VkDevice device)
    {
        std::vector<VkDescriptorSetLayout> layouts = {
                Camera::GetDescriptorSetLayout(),
                VolumeData::GetDescriptorSetLayout(),
                DirLight::GetDescriptorSetLayout(),
                PointLight::GetDescriptorSetLayout(),
                HdrEnvMap::GetDescriptorSetLayout(),
                m_DescSetLayout };

        VkPipelineLayoutCreateInfo layoutCreateInfo;
        layoutCreateInfo.sType = VK_STRUCTURE_TYPE_PIPELINE_LAYOUT_CREATE_INFO;
        layoutCreateInfo.pNext = nullptr;
        layoutCreateInfo.flags = 0;
        layoutCreateInfo.setLayoutCount = layouts.size();
        layoutCreateInfo.pSetLayouts = layouts.data();
        layoutCreateInfo.pushConstantRangeCount = 0;
        layoutCreateInfo.pPushConstantRanges = nullptr;

        VkResult result = vkCreatePipelineLayout(device, &layoutCreateInfo, nullptr, &m_PipelineLayout);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::InitSpecializationConstants()
    {
        // Fill struct
        m_SpecData.renderWidth = m_RenderWidth;
        m_SpecData.renderHeight = m_RenderHeight;
        m_SpecData.trainWidth = m_TrainWidth;
        m_SpecData.trainHeight = m_TrainHeight;
        m_SpecData.trainSpp = m_TrainSpp;

        m_SpecData.batchSize = m_Nrc.GetBatchSize();

        m_SpecData.volumeDensityFactor = m_HpmScene.GetVolumeData()->GetDensityFactor();
        m_SpecData.volumeG = m_HpmScene.GetVolumeData()->GetG();

        m_SpecData.hdrEnvMapStrength = m_HpmScene.GetHdrEnvMap()->GetStrength();

        // Init map entries
        uint32_t constantID = 0;

        VkSpecializationMapEntry renderWidthEntry;
        renderWidthEntry.constantID = constantID++;
        renderWidthEntry.offset = offsetof(SpecializationData, SpecializationData::renderWidth);
        renderWidthEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry renderHeightEntry;
        renderHeightEntry.constantID = constantID++;
        renderHeightEntry.offset = offsetof(SpecializationData, SpecializationData::renderHeight);
        renderHeightEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry trainWidthEntry;
        trainWidthEntry.constantID = constantID++;
        trainWidthEntry.offset = offsetof(SpecializationData, SpecializationData::trainWidth);
        trainWidthEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry trainHeightEntry;
        trainHeightEntry.constantID = constantID++;
        trainHeightEntry.offset = offsetof(SpecializationData, SpecializationData::trainHeight);
        trainHeightEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry trainSppEntry;
        trainSppEntry.constantID = constantID++;
        trainSppEntry.offset = offsetof(SpecializationData, SpecializationData::trainSpp);
        trainSppEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry batchSizeEntry;
        batchSizeEntry.constantID = constantID++;
        batchSizeEntry.offset = offsetof(SpecializationData, SpecializationData::batchSize);
        batchSizeEntry.size = sizeof(uint32_t);

        VkSpecializationMapEntry volumeDensityFactorEntry;
        volumeDensityFactorEntry.constantID = constantID++;
        volumeDensityFactorEntry.offset = offsetof(SpecializationData, SpecializationData::volumeDensityFactor);
        volumeDensityFactorEntry.size = sizeof(float);

        VkSpecializationMapEntry volumeGEntry;
        volumeGEntry.constantID = constantID++;
        volumeGEntry.offset = offsetof(SpecializationData, SpecializationData::volumeG);
        volumeGEntry.size = sizeof(float);

        VkSpecializationMapEntry hdrEnvMapStrengthEntry;
        hdrEnvMapStrengthEntry.constantID = constantID++;
        hdrEnvMapStrengthEntry.offset = offsetof(SpecializationData, SpecializationData::hdrEnvMapStrength);
        hdrEnvMapStrengthEntry.size = sizeof(float);

        m_SpecMapEntries = {
                renderWidthEntry,
                renderHeightEntry,
                trainWidthEntry,
                trainHeightEntry,
                trainSppEntry,
                batchSizeEntry,
                volumeDensityFactorEntry,
                volumeGEntry,
                hdrEnvMapStrengthEntry
        };

        m_SpecInfo.mapEntryCount = m_SpecMapEntries.size();
        m_SpecInfo.pMapEntries = m_SpecMapEntries.data();
        m_SpecInfo.dataSize = sizeof(SpecializationData);
        m_SpecInfo.pData = &m_SpecData;
    }

    void NrcHpmRenderer::CreateGenRaysPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_GenRaysShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_GenRaysPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrepInferRaysPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_PrepInferRaysShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_PrepInferRaysPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrepTrainRaysPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_PrepTrainRaysShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_PrepTrainRaysPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateRenderPipeline(VkDevice device)
    {
        VkPipelineShaderStageCreateInfo shaderStage;
        shaderStage.sType = VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
        shaderStage.pNext = nullptr;
        shaderStage.flags = 0;
        shaderStage.stage = VK_SHADER_STAGE_COMPUTE_BIT;
        shaderStage.module = m_RenderShader.GetVulkanModule();
        shaderStage.pName = "main";
        shaderStage.pSpecializationInfo = &m_SpecInfo;

        VkComputePipelineCreateInfo pipelineCI;
        pipelineCI.sType = VK_STRUCTURE_TYPE_COMPUTE_PIPELINE_CREATE_INFO;
        pipelineCI.pNext = nullptr;
        pipelineCI.flags = 0;
        pipelineCI.stage = shaderStage;
        pipelineCI.layout = m_PipelineLayout;
        pipelineCI.basePipelineHandle = VK_NULL_HANDLE;
        pipelineCI.basePipelineIndex = 0;

        VkResult result = vkCreateComputePipelines(device, VK_NULL_HANDLE, 1, &pipelineCI, nullptr, &m_RenderPipeline);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateOutputImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_SAMPLED_BIT | VK_IMAGE_USAGE_STORAGE_BIT | VK_IMAGE_USAGE_TRANSFER_SRC_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_OutputImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_OutputImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_OutputImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_OutputImage, m_OutputImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_OutputImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_OutputImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_RandomTasksCmdBuf, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_RandomTasksCmdBuf,
                m_OutputImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_RandomTasksCmdBuf);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrimaryRayColorImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_PrimaryRayColorImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_PrimaryRayColorImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_PrimaryRayColorImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_PrimaryRayColorImage, m_PrimaryRayColorImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_PrimaryRayColorImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_PrimaryRayColorImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_RandomTasksCmdBuf, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_RandomTasksCmdBuf,
                m_PrimaryRayColorImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_RandomTasksCmdBuf);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreatePrimaryRayInfoImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_PrimaryRayInfoImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_PrimaryRayInfoImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_PrimaryRayInfoImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_PrimaryRayInfoImage, m_PrimaryRayInfoImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_PrimaryRayInfoImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_PrimaryRayInfoImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_RandomTasksCmdBuf, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_RandomTasksCmdBuf,
                m_PrimaryRayInfoImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_RandomTasksCmdBuf);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateNrcRayOriginImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_NrcRayOriginImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_NrcRayOriginImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_NrcRayOriginImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_NrcRayOriginImage, m_NrcRayOriginImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_NrcRayOriginImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_NrcRayOriginImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_RandomTasksCmdBuf, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_RandomTasksCmdBuf,
                m_NrcRayOriginImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_RandomTasksCmdBuf);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::CreateNrcRayDirImage(VkDevice device)
    {
        VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;

        // Create Image
        VkImageCreateInfo imageCI;
        imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
        imageCI.pNext = nullptr;
        imageCI.flags = 0;
        imageCI.imageType = VK_IMAGE_TYPE_2D;
        imageCI.format = format;
        imageCI.extent = { m_RenderWidth, m_RenderHeight, 1 };
        imageCI.mipLevels = 1;
        imageCI.arrayLayers = 1;
        imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
        imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
        imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT;
        imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
        imageCI.queueFamilyIndexCount = 0;
        imageCI.pQueueFamilyIndices = nullptr;
        imageCI.initialLayout = VK_IMAGE_LAYOUT_PREINITIALIZED;

        VkResult result = vkCreateImage(device, &imageCI, nullptr, &m_NrcRayDirImage);
        ASSERT_VULKAN(result);

        // Image Memory
        VkMemoryRequirements memoryRequirements;
        vkGetImageMemoryRequirements(device, m_NrcRayDirImage, &memoryRequirements);

        VkMemoryAllocateInfo allocateInfo;
        allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
        allocateInfo.pNext = nullptr;
        allocateInfo.allocationSize = memoryRequirements.size;
        allocateInfo.memoryTypeIndex = VulkanAPI::FindMemoryType(
                memoryRequirements.memoryTypeBits,
                VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

        result = vkAllocateMemory(device, &allocateInfo, nullptr, &m_NrcRayDirImageMemory);
        ASSERT_VULKAN(result);

        result = vkBindImageMemory(device, m_NrcRayDirImage, m_NrcRayDirImageMemory, 0);
        ASSERT_VULKAN(result);

        // Create image view
        VkImageViewCreateInfo imageViewCI;
        imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
        imageViewCI.pNext = nullptr;
        imageViewCI.flags = 0;
        imageViewCI.image = m_NrcRayDirImage;
        imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
        imageViewCI.format = format;
        imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
        imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageViewCI.subresourceRange.baseMipLevel = 0;
        imageViewCI.subresourceRange.levelCount = 1;
        imageViewCI.subresourceRange.baseArrayLayer = 0;
        imageViewCI.subresourceRange.layerCount = 1;

        result = vkCreateImageView(device, &imageViewCI, nullptr, &m_NrcRayDirImageView);
        ASSERT_VULKAN(result);

        // Change image layout
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        result = vkBeginCommandBuffer(m_RandomTasksCmdBuf, &beginInfo);
        ASSERT_VULKAN(result);

        vk::CommandRecorder::ImageLayoutTransfer(
                m_RandomTasksCmdBuf,
                m_NrcRayDirImage,
                VK_IMAGE_LAYOUT_PREINITIALIZED,
                VK_IMAGE_LAYOUT_GENERAL,
                VK_ACCESS_NONE,
                VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
                VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

        result = vkEndCommandBuffer(m_RandomTasksCmdBuf);
        ASSERT_VULKAN(result);

        VkSubmitInfo submitInfo;
        submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
        submitInfo.pNext = nullptr;
        submitInfo.waitSemaphoreCount = 0;
        submitInfo.pWaitSemaphores = nullptr;
        submitInfo.pWaitDstStageMask = nullptr;
        submitInfo.commandBufferCount = 1;
        submitInfo.pCommandBuffers = &m_RandomTasksCmdBuf;
        submitInfo.signalSemaphoreCount = 0;
        submitInfo.pSignalSemaphores = nullptr;

        VkQueue queue = VulkanAPI::GetGraphicsQueue();
        result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
        ASSERT_VULKAN(result);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::AllocateAndUpdateDescriptorSet(VkDevice device)
    {
        // Allocate
        VkDescriptorSetAllocateInfo descSetAI;
        descSetAI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
        descSetAI.pNext = nullptr;
        descSetAI.descriptorPool = m_DescPool;
        descSetAI.descriptorSetCount = 1;
        descSetAI.pSetLayouts = &m_DescSetLayout;

        VkResult result = vkAllocateDescriptorSets(device, &descSetAI, &m_DescSet);
        ASSERT_VULKAN(result);

        // Write
        // Storage image writes
        uint32_t bindingIndex = 0;

        VkDescriptorImageInfo outputImageInfo;
        outputImageInfo.sampler = VK_NULL_HANDLE;
        outputImageInfo.imageView = m_OutputImageView;
        outputImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet outputImageWrite;
        outputImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        outputImageWrite.pNext = nullptr;
        outputImageWrite.dstSet = m_DescSet;
        outputImageWrite.dstBinding = bindingIndex++;
        outputImageWrite.dstArrayElement = 0;
        outputImageWrite.descriptorCount = 1;
        outputImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        outputImageWrite.pImageInfo = &outputImageInfo;
        outputImageWrite.pBufferInfo = nullptr;
        outputImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo primaryRayColorImageInfo;
        primaryRayColorImageInfo.sampler = VK_NULL_HANDLE;
        primaryRayColorImageInfo.imageView = m_PrimaryRayColorImageView;
        primaryRayColorImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet primaryRayColorImageWrite;
        primaryRayColorImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        primaryRayColorImageWrite.pNext = nullptr;
        primaryRayColorImageWrite.dstSet = m_DescSet;
        primaryRayColorImageWrite.dstBinding = bindingIndex++;
        primaryRayColorImageWrite.dstArrayElement = 0;
        primaryRayColorImageWrite.descriptorCount = 1;
        primaryRayColorImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayColorImageWrite.pImageInfo = &primaryRayColorImageInfo;
        primaryRayColorImageWrite.pBufferInfo = nullptr;
        primaryRayColorImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo primaryRayInfoImageInfo;
        primaryRayInfoImageInfo.sampler = VK_NULL_HANDLE;
        primaryRayInfoImageInfo.imageView = m_PrimaryRayInfoImageView;
        primaryRayInfoImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet primaryRayInfoImageWrite;
        primaryRayInfoImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        primaryRayInfoImageWrite.pNext = nullptr;
        primaryRayInfoImageWrite.dstSet = m_DescSet;
        primaryRayInfoImageWrite.dstBinding = bindingIndex++;
        primaryRayInfoImageWrite.dstArrayElement = 0;
        primaryRayInfoImageWrite.descriptorCount = 1;
        primaryRayInfoImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        primaryRayInfoImageWrite.pImageInfo = &primaryRayInfoImageInfo;
        primaryRayInfoImageWrite.pBufferInfo = nullptr;
        primaryRayInfoImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo nrcRayOriginImageInfo;
        nrcRayOriginImageInfo.sampler = VK_NULL_HANDLE;
        nrcRayOriginImageInfo.imageView = m_NrcRayOriginImageView;
        nrcRayOriginImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet nrcRayOriginImageWrite;
        nrcRayOriginImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcRayOriginImageWrite.pNext = nullptr;
        nrcRayOriginImageWrite.dstSet = m_DescSet;
        nrcRayOriginImageWrite.dstBinding = bindingIndex++;
        nrcRayOriginImageWrite.dstArrayElement = 0;
        nrcRayOriginImageWrite.descriptorCount = 1;
        nrcRayOriginImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayOriginImageWrite.pImageInfo = &nrcRayOriginImageInfo;
        nrcRayOriginImageWrite.pBufferInfo = nullptr;
        nrcRayOriginImageWrite.pTexelBufferView = nullptr;

        VkDescriptorImageInfo nrcRayDirImageInfo;
        nrcRayDirImageInfo.sampler = VK_NULL_HANDLE;
        nrcRayDirImageInfo.imageView = m_NrcRayDirImageView;
        nrcRayDirImageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

        VkWriteDescriptorSet nrcRayDirImageWrite;
        nrcRayDirImageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcRayDirImageWrite.pNext = nullptr;
        nrcRayDirImageWrite.dstSet = m_DescSet;
        nrcRayDirImageWrite.dstBinding = bindingIndex++;
        nrcRayDirImageWrite.dstArrayElement = 0;
        nrcRayDirImageWrite.descriptorCount = 1;
        nrcRayDirImageWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_IMAGE;
        nrcRayDirImageWrite.pImageInfo = &nrcRayDirImageInfo;
        nrcRayDirImageWrite.pBufferInfo = nullptr;
        nrcRayDirImageWrite.pTexelBufferView = nullptr;

        // Storage buffer writes
        VkDescriptorBufferInfo nrcInferInputBufferInfo;
        nrcInferInputBufferInfo.buffer = m_NrcInferInputBuffer->GetVulkanHandle();
        nrcInferInputBufferInfo.offset = 0;
        nrcInferInputBufferInfo.range = m_NrcInferInputBufferSize;

        VkWriteDescriptorSet nrcInferInputBufferWrite;
        nrcInferInputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcInferInputBufferWrite.pNext = nullptr;
        nrcInferInputBufferWrite.dstSet = m_DescSet;
        nrcInferInputBufferWrite.dstBinding = bindingIndex++;
        nrcInferInputBufferWrite.dstArrayElement = 0;
        nrcInferInputBufferWrite.descriptorCount = 1;
        nrcInferInputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferInputBufferWrite.pImageInfo = nullptr;
        nrcInferInputBufferWrite.pBufferInfo = &nrcInferInputBufferInfo;
        nrcInferInputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcInferOutputBufferInfo;
        nrcInferOutputBufferInfo.buffer = m_NrcInferOutputBuffer->GetVulkanHandle();
        nrcInferOutputBufferInfo.offset = 0;
        nrcInferOutputBufferInfo.range = m_NrcInferOutputBufferSize;

        VkWriteDescriptorSet nrcInferOutputBufferWrite;
        nrcInferOutputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcInferOutputBufferWrite.pNext = nullptr;
        nrcInferOutputBufferWrite.dstSet = m_DescSet;
        nrcInferOutputBufferWrite.dstBinding = bindingIndex++;
        nrcInferOutputBufferWrite.dstArrayElement = 0;
        nrcInferOutputBufferWrite.descriptorCount = 1;
        nrcInferOutputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferOutputBufferWrite.pImageInfo = nullptr;
        nrcInferOutputBufferWrite.pBufferInfo = &nrcInferOutputBufferInfo;
        nrcInferOutputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcTrainInputBufferInfo;
        nrcTrainInputBufferInfo.buffer = m_NrcTrainInputBuffer->GetVulkanHandle();
        nrcTrainInputBufferInfo.offset = 0;
        nrcTrainInputBufferInfo.range = m_NrcTrainInputBufferSize;

        VkWriteDescriptorSet nrcTrainInputBufferWrite;
        nrcTrainInputBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcTrainInputBufferWrite.pNext = nullptr;
        nrcTrainInputBufferWrite.dstSet = m_DescSet;
        nrcTrainInputBufferWrite.dstBinding = bindingIndex++;
        nrcTrainInputBufferWrite.dstArrayElement = 0;
        nrcTrainInputBufferWrite.descriptorCount = 1;
        nrcTrainInputBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainInputBufferWrite.pImageInfo = nullptr;
        nrcTrainInputBufferWrite.pBufferInfo = &nrcTrainInputBufferInfo;
        nrcTrainInputBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcTrainTargetBufferInfo;
        nrcTrainTargetBufferInfo.buffer = m_NrcTrainTargetBuffer->GetVulkanHandle();
        nrcTrainTargetBufferInfo.offset = 0;
        nrcTrainTargetBufferInfo.range = m_NrcTrainTargetBufferSize;

        VkWriteDescriptorSet nrcTrainTargetBufferWrite;
        nrcTrainTargetBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcTrainTargetBufferWrite.pNext = nullptr;
        nrcTrainTargetBufferWrite.dstSet = m_DescSet;
        nrcTrainTargetBufferWrite.dstBinding = bindingIndex++;
        nrcTrainTargetBufferWrite.dstArrayElement = 0;
        nrcTrainTargetBufferWrite.descriptorCount = 1;
        nrcTrainTargetBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainTargetBufferWrite.pImageInfo = nullptr;
        nrcTrainTargetBufferWrite.pBufferInfo = &nrcTrainTargetBufferInfo;
        nrcTrainTargetBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcInferFilterBufferInfo;
        nrcInferFilterBufferInfo.buffer = m_NrcInferFilterBuffer->GetVulkanHandle();
        nrcInferFilterBufferInfo.offset = 0;
        nrcInferFilterBufferInfo.range = m_NrcInferFilterBufferSize;

        VkWriteDescriptorSet nrcInferFilterBufferWrite;
        nrcInferFilterBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcInferFilterBufferWrite.pNext = nullptr;
        nrcInferFilterBufferWrite.dstSet = m_DescSet;
        nrcInferFilterBufferWrite.dstBinding = bindingIndex++;
        nrcInferFilterBufferWrite.dstArrayElement = 0;
        nrcInferFilterBufferWrite.descriptorCount = 1;
        nrcInferFilterBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcInferFilterBufferWrite.pImageInfo = nullptr;
        nrcInferFilterBufferWrite.pBufferInfo = &nrcInferFilterBufferInfo;
        nrcInferFilterBufferWrite.pTexelBufferView = nullptr;

        VkDescriptorBufferInfo nrcTrainRayResBufferInfo;
        nrcTrainRayResBufferInfo.buffer = m_NrcTrainRayResBuffer->GetVulkanHandle();
        nrcTrainRayResBufferInfo.offset = 0;
        nrcTrainRayResBufferInfo.range = m_NrcTrainRayResBufferSize;

        VkWriteDescriptorSet nrcTrainRayResBufferWrite;
        nrcTrainRayResBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        nrcTrainRayResBufferWrite.pNext = nullptr;
        nrcTrainRayResBufferWrite.dstSet = m_DescSet;
        nrcTrainRayResBufferWrite.dstBinding = bindingIndex++;
        nrcTrainRayResBufferWrite.dstArrayElement = 0;
        nrcTrainRayResBufferWrite.descriptorCount = 1;
        nrcTrainRayResBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_STORAGE_BUFFER;
        nrcTrainRayResBufferWrite.pImageInfo = nullptr;
        nrcTrainRayResBufferWrite.pBufferInfo = &nrcTrainRayResBufferInfo;
        nrcTrainRayResBufferWrite.pTexelBufferView = nullptr;

        // Uniform buffer write
        VkDescriptorBufferInfo uniformBufferInfo;
        uniformBufferInfo.buffer = m_UniformBuffer.GetVulkanHandle();
        uniformBufferInfo.offset = 0;
        uniformBufferInfo.range = sizeof(UniformData);

        VkWriteDescriptorSet uniformBufferWrite;
        uniformBufferWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        uniformBufferWrite.pNext = nullptr;
        uniformBufferWrite.dstSet = m_DescSet;
        uniformBufferWrite.dstBinding = bindingIndex++;
        uniformBufferWrite.dstArrayElement = 0;
        uniformBufferWrite.descriptorCount = 1;
        uniformBufferWrite.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
        uniformBufferWrite.pImageInfo = nullptr;
        uniformBufferWrite.pBufferInfo = &uniformBufferInfo;
        uniformBufferWrite.pTexelBufferView = nullptr;

        // Write writes
        std::vector<VkWriteDescriptorSet> writes = {
                outputImageWrite,
                primaryRayColorImageWrite,
                primaryRayInfoImageWrite,
                nrcRayOriginImageWrite,
                nrcRayDirImageWrite,
                nrcInferInputBufferWrite,
                nrcInferOutputBufferWrite,
                nrcTrainInputBufferWrite,
                nrcTrainTargetBufferWrite,
                nrcInferFilterBufferWrite,
                nrcTrainRayResBufferWrite,
                uniformBufferWrite
        };

        vkUpdateDescriptorSets(device, writes.size(), writes.data(), 0, nullptr);
    }

    void NrcHpmRenderer::CreateQueryPool(VkDevice device)
    {
        VkQueryPoolCreateInfo queryPoolCI;
        queryPoolCI.sType = VK_STRUCTURE_TYPE_QUERY_POOL_CREATE_INFO;
        queryPoolCI.pNext = nullptr;
        queryPoolCI.flags = 0;
        queryPoolCI.queryType = VK_QUERY_TYPE_TIMESTAMP;
        queryPoolCI.queryCount = c_QueryCount;
        queryPoolCI.pipelineStatistics = 0;

        ASSERT_VULKAN(vkCreateQueryPool(device, &queryPoolCI, nullptr, &m_QueryPool));
    }

    void NrcHpmRenderer::RecordPreCudaCommandBuffer()
    {
        m_QueryIndex = 0;

        // Begin
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        VkResult result = vkBeginCommandBuffer(m_PreCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        // Reset query pool
        vkCmdResetQueryPool(m_PreCudaCommandBuffer, m_QueryPool, 0, c_QueryCount);

        // Clear buffers
        vkCmdFillBuffer(m_PreCudaCommandBuffer, m_NrcInferInputBuffer->GetVulkanHandle(), 0, VK_WHOLE_SIZE, 0);
        vkCmdFillBuffer(m_PreCudaCommandBuffer, m_NrcInferOutputBuffer->GetVulkanHandle(), 0, VK_WHOLE_SIZE, 0);
        vkCmdFillBuffer(m_PreCudaCommandBuffer, m_NrcTrainInputBuffer->GetVulkanHandle(), 0, VK_WHOLE_SIZE, 0);
        vkCmdFillBuffer(m_PreCudaCommandBuffer, m_NrcTrainTargetBuffer->GetVulkanHandle(), 0, VK_WHOLE_SIZE, 0);
        vkCmdFillBuffer(m_PreCudaCommandBuffer, m_NrcInferFilterBuffer->GetVulkanHandle(), 0, VK_WHOLE_SIZE, 0);

        // Collect descriptor sets
        std::vector<VkDescriptorSet> descSets = { m_Camera->GetDescriptorSet() };
        const std::vector<VkDescriptorSet>& hpmSceneDescSets = m_HpmScene.GetDescriptorSets();
        descSets.insert(descSets.end(), hpmSceneDescSets.begin(), hpmSceneDescSets.end());
        descSets.push_back(m_DescSet);

        // Create memory barrier
        VkMemoryBarrier memoryBarrier;
        memoryBarrier.sType = VK_STRUCTURE_TYPE_MEMORY_BARRIER;
        memoryBarrier.pNext = nullptr;
        memoryBarrier.srcAccessMask = VK_ACCESS_MEMORY_WRITE_BIT;
        memoryBarrier.dstAccessMask = VK_ACCESS_MEMORY_READ_BIT;

        // Bind descriptor sets
        vkCmdBindDescriptorSets(
                m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PipelineLayout,
                0, descSets.size(), descSets.data(),
                0, nullptr);

        // Timestamp
        vkCmdWriteTimestamp(m_PreCudaCommandBuffer, VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT, m_QueryPool, m_QueryIndex++);

        // Gen rays pipeline
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_GenRaysPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        vkCmdPipelineBarrier(
                m_PreCudaCommandBuffer,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_DEPENDENCY_DEVICE_GROUP_BIT,
                1, &memoryBarrier,
                0, nullptr,
                0, nullptr);

        // Timestamp
        vkCmdWriteTimestamp(m_PreCudaCommandBuffer, VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT, m_QueryPool, m_QueryIndex++);

        // Prep infer rays
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PrepInferRaysPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        vkCmdPipelineBarrier(
                m_PreCudaCommandBuffer,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT,
                VK_DEPENDENCY_DEVICE_GROUP_BIT,
                1, &memoryBarrier,
                0, nullptr,
                0, nullptr);

        // Timestamp
        vkCmdWriteTimestamp(m_PreCudaCommandBuffer, VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT, m_QueryPool, m_QueryIndex++);

        // Prep train rays
        vkCmdBindPipeline(m_PreCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PrepTrainRaysPipeline);
        vkCmdDispatch(m_PreCudaCommandBuffer, m_TrainWidth / 32, m_TrainHeight, 1);

        // Timestamp
        vkCmdWriteTimestamp(m_PreCudaCommandBuffer, VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT, m_QueryPool, m_QueryIndex++);

        // End
        result = vkEndCommandBuffer(m_PreCudaCommandBuffer);
        ASSERT_VULKAN(result);
    }

    void NrcHpmRenderer::RecordPostCudaCommandBuffer()
    {
        // Begin command buffer
        VkCommandBufferBeginInfo beginInfo;
        beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
        beginInfo.pNext = nullptr;
        beginInfo.flags = 0;
        beginInfo.pInheritanceInfo = nullptr;

        VkResult result = vkBeginCommandBuffer(m_PostCudaCommandBuffer, &beginInfo);
        ASSERT_VULKAN(result);

        // Collect descriptor sets
        std::vector<VkDescriptorSet> descSets = { m_Camera->GetDescriptorSet() };
        const std::vector<VkDescriptorSet>& hpmSceneDescSets = m_HpmScene.GetDescriptorSets();
        descSets.insert(descSets.end(), hpmSceneDescSets.begin(), hpmSceneDescSets.end());
        descSets.push_back(m_DescSet);

        // Bind descriptor sets
        vkCmdBindDescriptorSets(
                m_PostCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_PipelineLayout,
                0, descSets.size(), descSets.data(),
                0, nullptr);

        // Timestamp
        vkCmdWriteTimestamp(m_PostCudaCommandBuffer, VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT, m_QueryPool, m_QueryIndex++);

        // Render pipeline
        vkCmdBindPipeline(m_PostCudaCommandBuffer, VK_PIPELINE_BIND_POINT_COMPUTE, m_RenderPipeline);
        vkCmdDispatch(m_PostCudaCommandBuffer, m_RenderWidth / 32, m_RenderHeight, 1);

        // Timestamp
        vkCmdWriteTimestamp(m_PostCudaCommandBuffer, VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT, m_QueryPool, m_QueryIndex++);

        // End command buffer
        result = vkEndCommandBuffer(m_PostCudaCommandBuffer);
        ASSERT_VULKAN(result);
    }
}
