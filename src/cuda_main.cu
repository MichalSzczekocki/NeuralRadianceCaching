//#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

en::NrcHpmRenderer* hpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (hpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");

    //uint32_t width = en::Window::GetWidth();
    //uint32_t height = en::Window::GetHeight();
    //nrcHpmRenderer->ResizeFrame(width, height);
    //en::ImGuiRenderer::Resize(width, height);
    //en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

int main()
{
    // Start engine
    const std::string appName("NeuralRadianceCaching");
    uint32_t width = 768; // Multiple of 128 for nrc batch size
    uint32_t height = width;
    en::Log::Info("Starting " + appName);
    en::Window::Init(width, height, false, appName);
    en::Input::Init(en::Window::GetGLFWHandle());
    en::VulkanAPI::Init(appName);
    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    // Init nrc
    nlohmann::json config = {
            {"loss", {
                             {"otype", "L2"}
                     }},
            {"optimizer", {
                             {"otype", "Adam"},
                             {"learning_rate", 1e-3},
                     }},
            {"encoding", {
                             {"otype", "Composite"},
                             {"reduction", "Concatenation"},
                             {"nested", {
                                                {
                                                        {"otype", "HashGrid"},
                                                        {"n_dims_to_encode", 3},
                                                        {"n_levels", 16},
                                                        {"n_features_per_level", 2},
                                                        {"log2_hashmap_size", 19},
                                                        {"base_resolution", 16},
                                                        {"per_level_scale", 2.0},
                                                },
                                                {
                                                        {"otype", "OneBlob"},
                                                        {"n_dims_to_encode", 2},
                                                        {"n_bins", 4},
                                                },
                                        }},
                     }},
            {"network", {
                             {"otype", "FullyFusedMLP"},
                             {"activation", "ReLU"},
                             {"output_activation", "None"},
                             {"n_neurons", 128},
                             {"n_hidden_layers", 6},
                     }},
    };

    en::NeuralRadianceCache nrc(config, 5, 3, 14);

    // Lighting
    en::DirLight dirLight(-1.57f, 0.0f, glm::vec3(1.0f), 1.5f);
    en::PointLight pointLight(glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(1.0f, 1.0f, 1.0f), 0.0f);

    int hdrWidth, hdrHeight;
    std::vector<float> hdr4fData = en::ReadFileHdr4f("data/image/mountain.hdr", hdrWidth, hdrHeight);
    std::array<std::vector<float>, 2> hdrCdf = en::Hdr4fToCdf(hdr4fData, hdrWidth, hdrHeight);
    en::HdrEnvMap hdrEnvMap(
            1.0f,
            3.0f,
            hdrWidth,
            hdrHeight,
            hdr4fData,
            hdrCdf[0],
            hdrCdf[1]);

    // Load data
    auto density3D = en::ReadFileDensity3D("data/cloud_sixteenth", 125, 85, 153);
    en::vk::Texture3D density3DTex(
            density3D,
            VK_FILTER_LINEAR,
            VK_SAMPLER_ADDRESS_MODE_CLAMP_TO_BORDER,
            VK_BORDER_COLOR_INT_OPAQUE_BLACK);
    en::VolumeData volumeData(&density3DTex);

    // Setup rendering
    en::Camera camera(
            glm::vec3(0.0f, 0.0f, -64.0f),
            glm::vec3(0.0f, 0.0f, 1.0f),
            glm::vec3(0.0f, 1.0f, 0.0f),
            static_cast<float>(width) / static_cast<float>(height),
            glm::radians(60.0f),
            0.1f,
            100.0f);

    // Init rendering pipeline
    en::vk::Swapchain swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);

    hpmRenderer = new en::NrcHpmRenderer(
            width,
            height,
            128,
            128,
            camera,
            volumeData,
            dirLight,
            pointLight,
            hdrEnvMap,
            nrc);

    en::ImGuiRenderer::Init(width, height);
    en::ImGuiRenderer::SetBackgroundImageView(hpmRenderer->GetImageView());

    // Swapchain rerecording because imgui renderer is now available
    swapchain.Resize(width, height);

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    while (!en::Window::IsClosed())
    {
        // Update
        en::Window::Update();
        en::Input::Update();
        en::Time::Update();

        width = en::Window::GetWidth();
        height = en::Window::GetHeight();

        float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
        uint32_t fps = en::Time::GetFps();
        en::Window::SetTitle(appName + " | Delta time: " + std::to_string(deltaTime) + "s | Fps: " + std::to_string(fps));

        // Physics
        en::Input::HandleUserCamInput(&camera, deltaTime);
        camera.SetAspectRatio(width, height);
        camera.UpdateUniformBuffer();

        // Render
        hpmRenderer->Render(queue);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        // Imgui
        en::ImGuiRenderer::StartFrame();

        volumeData.RenderImGui();
        volumeData.Update(camera.HasChanged());
        dirLight.RenderImgui();
        pointLight.RenderImGui();
        hdrEnvMap.RenderImGui();

        en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        // Display
        swapchain.DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE);
        frameCount++;
    }
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    en::ImGuiRenderer::Shutdown();

    swapchain.Destroy(true);

    en::VulkanAPI::Shutdown();
    en::Window::Shutdown();

    en::Log::Info("Ending " + appName);

    return 0;
}
