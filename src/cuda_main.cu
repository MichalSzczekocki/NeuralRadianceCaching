//#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

en::NrcHpmRenderer* hpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (hpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");

    //uint32_t width = en::Window::GetWidth();
    //uint32_t height = en::Window::GetHeight();
    //nrcHpmRenderer->ResizeFrame(width, height);
    //en::ImGuiRenderer::Resize(width, height);
    //en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
    // Start engine
    const std::string appName("NeuralRadianceCaching");
    uint32_t width = appConfig.renderWidth;
    uint32_t height = appConfig.renderHeight;
    en::Log::Info("Starting " + appName);
    en::Window::Init(width, height, false, appName);
    en::Input::Init(en::Window::GetGLFWHandle());
    en::VulkanAPI::Init(appName);

    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    // Init resources
    en::NeuralRadianceCache nrc(appConfig);

    en::HpmScene hpmScene(appConfig);

    // Setup rendering
    en::Camera camera(
            glm::vec3(0.0f, 0.0f, -64.0f),
            glm::vec3(0.0f, 0.0f, 1.0f),
            glm::vec3(0.0f, 1.0f, 0.0f),
            static_cast<float>(width) / static_cast<float>(height),
            glm::radians(60.0f),
            0.1f,
            100.0f);

    // Init rendering pipeline
    en::vk::Swapchain swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);

    hpmRenderer = new en::NrcHpmRenderer(
            width,
            height,
            0.05f,
            1,
            camera,
            hpmScene,
            nrc);

    en::ImGuiRenderer::Init(width, height);
    en::ImGuiRenderer::SetBackgroundImageView(hpmRenderer->GetImageView());

    // Swapchain rerecording because imgui renderer is now available
    swapchain.Resize(width, height);

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    bool shutdown = false;
    bool restartAfterClose = false;
    while (!en::Window::IsClosed() && !shutdown)
    {
        // Exit
        if (frameCount == 1000) { break; }

        // Update
        en::Window::Update();
        en::Input::Update();
        en::Time::Update();

        width = en::Window::GetWidth();
        height = en::Window::GetHeight();

        float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
        uint32_t fps = en::Time::GetFps();
        en::Window::SetTitle(appName + " | Delta time: " + std::to_string(deltaTime) + "s | Fps: " + std::to_string(fps));

        if (frameCount % 100 == 0)
        {
            en::Log::Info("Loss: " + std::to_string(nrc.GetLoss()));
        }

        // Physics
        en::Input::HandleUserCamInput(&camera, deltaTime);
        camera.SetAspectRatio(width, height);
        camera.UpdateUniformBuffer();

        // Render
        hpmRenderer->Render(queue);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        // Imgui
        en::ImGuiRenderer::StartFrame();
        ImGui::Begin("Statistics");
        ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
        ImGui::Text("DeltaTime %f", deltaTime);
        ImGui::Text("FPS %d", fps);
        ImGui::Text("NRC Loss %f", nrc.GetLoss());
        ImGui::End();

        ImGui::Begin("Controls");
        shutdown = ImGui::Button("Shutdown");
        ImGui::Checkbox("Restart after shutdown", &restartAfterClose);
        ImGui::End();

        hpmScene.Update(true);

        appConfig.RenderImGui();

        en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        // Display
        swapchain.DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE);
        frameCount++;
    }
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    hpmRenderer->Destroy();
    delete hpmRenderer;

    en::ImGuiRenderer::Shutdown();

    swapchain.Destroy(true);

    hpmScene.Destroy();
    camera.Destroy();
    nrc.Destroy();

    en::VulkanAPI::Shutdown();
    en::Window::Shutdown();

    en::Log::Info("Ending " + appName);

    return restartAfterClose;
}

int main(int argc, char** argv)
{
    en::AppConfig appConfig(argc, argv);

    bool restartRunConfig;
    do {
        restartRunConfig = RunAppConfigInstance(appConfig);
    } while (restartRunConfig);

    return 0;
}
