//#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <filesystem>
#include <engine/graphics/renderer/McHpmRenderer.hpp>
#include <tinyexr.h>
#include <engine/graphics/vulkan/CommandPool.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>
en::McHpmRenderer* gtRenderer = nullptr;
std::array<en::Camera*, 6> testCameras = { nullptr, nullptr, nullptr, nullptr, nullptr, nullptr };
std::array<float*, 6> gtImages = { nullptr, nullptr, nullptr, nullptr, nullptr, nullptr };

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");
}

void Benchmark(
        uint32_t width,
        uint32_t height,
        uint32_t sceneID,
        const en::AppConfig& appConfig,
        const en::HpmScene& scene,
        const en::Camera* oldCamera,
        VkQueue queue)
{
    // Create output path if not exists
    std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
    if (!std::filesystem::is_directory(outputDirPath) || !std::filesystem::exists(outputDirPath))
    {
        std::filesystem::create_directory(outputDirPath);
    }

    // Create reference folder if not exists
    std::string referenceDirPath = "output/" + std::to_string(sceneID) + "/";
    if (!std::filesystem::is_directory(referenceDirPath) || !std::filesystem::exists(referenceDirPath))
    {
        en::Log::Info("Reference folder for scene " + std::to_string(sceneID) + " was not found. Creating reference images");

        // Create folder
        std::filesystem::create_directory(referenceDirPath);

        // Generate reference data
        for (size_t i = 0; i < testCameras.size(); i++)
        {
            en::Log::Info("Generating reference image " + std::to_string(i));

            // Set new camera
            gtRenderer->SetCamera(queue, testCameras[i]);

            // Generate reference image
            for (size_t frame = 0; frame < 8192; frame++)
            {
                gtRenderer->Render(queue);
                ASSERT_VULKAN(vkQueueWaitIdle(queue));
            }

            // Export reference image
            gtRenderer->ExportOutputImageToFile(queue, referenceDirPath + std::to_string(i) + ".exr");
        }
    }

    // Load reference images from folder
    for (size_t i = 0; i < testCameras.size(); i++)
    {
        if (gtImages[i] != nullptr) { continue; }
        int exrWidth;
        int exrHeight;

        const std::string exrFilePath = referenceDirPath + std::to_string(i) + ".exr";
        if (TINYEXR_SUCCESS != LoadEXR(&gtImages[i], &exrWidth, &exrHeight, exrFilePath.c_str(), nullptr))
        {
            en::Log::Error("Tinyexr failed to load " + exrFilePath, true);
        }

        if (exrWidth != width || exrHeight != height)
        {
            en::Log::Error("Extent of loaded reference image does not match renderer extent", true);
        }
    }

    // Test frame
    const bool prevNrcBlend = nrcHpmRenderer->IsBlending();
    const bool prevMcBlend = mcHpmRenderer->IsBlending();
    nrcHpmRenderer->SetBlend(true);
    mcHpmRenderer->SetBlend(true);

    std::array<float, testCameras.size()> nrcMseLosses;
    std::array<float, testCameras.size()> mcMseLosses;
    for (size_t i = 0; i < testCameras.size(); i++)
    {
        nrcHpmRenderer->SetCamera(queue, testCameras[i]);
        mcHpmRenderer->SetCamera(queue, testCameras[i]);
        for (size_t frame = 0; frame < 1; frame++)
        {
            nrcHpmRenderer->Render(queue);
            ASSERT_VULKAN(vkQueueWaitIdle(queue));

            mcHpmRenderer->Render(queue);
            ASSERT_VULKAN(vkQueueWaitIdle(queue));
        }
        nrcMseLosses[i] = nrcHpmRenderer->CompareReferenceMSE(queue, gtImages[i]);
        mcMseLosses[i] = mcHpmRenderer->CompareReferenceMSE(queue, gtImages[i]);
    }

    nrcHpmRenderer->SetBlend(prevNrcBlend);
    mcHpmRenderer->SetBlend(prevMcBlend);

    // Calculate total loss
    float nrcMSE = 0.0f;
    float mcMSE = 0.0f;
    for (size_t i = 0; i < testCameras.size(); i++)
    {
        nrcMSE += nrcMseLosses[i];
        mcMSE += mcMseLosses[i];
    }
    const float frameCountF = static_cast<float>(testCameras.size());
    en::Log::Info("NRC MSE: " + std::to_string(nrcMSE / frameCountF));
    en::Log::Info("MC MSE: " + std::to_string(mcMSE / frameCountF));

    // Reset camera
    nrcHpmRenderer->SetCamera(queue, oldCamera);
    mcHpmRenderer->SetCamera(queue, oldCamera);
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
    // Start engine
    const std::string appName("NeuralRadianceCaching");
    uint32_t width = appConfig.renderWidth;
    uint32_t height = appConfig.renderHeight;
    en::Log::Info("Starting " + appName);

    en::Window::Init(width, height, false, appName);
    if (en::Window::IsSupported()) { en::Input::Init(en::Window::GetGLFWHandle()); }
    en::VulkanAPI::Init(appName);
    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    // Renderer select
    const std::vector<char*> rendererMenuItems = { "MC", "NRC" }; // TODO: Restir
    const char* currentRendererMenuItem = rendererMenuItems[1];
    uint32_t rendererId = 1;

    // Init resources
    en::NeuralRadianceCache nrc(appConfig);

    en::HpmScene hpmScene(appConfig);

    // Setup rendering
    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);
    en::Camera camera(
            glm::vec3(64.0f, 0.0f, 0.0f),
            glm::vec3(-1.0f, 0.0f, 0.0f),
            glm::vec3(0.0f, 1.0f, 0.0f),
            aspectRatio,
            glm::radians(60.0f),
            0.1f,
            100.0f);

    testCameras = {
            new en::Camera(
                    glm::vec3(64.0f, 0.0f, 0.0f),
                    glm::vec3(-1.0f, 0.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            new	en::Camera(
                    glm::vec3(-64.0f, 0.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            new en::Camera(
                    glm::vec3(0.0f, 64.0f, 0.0f),
                    glm::vec3(0.0f, -1.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            new en::Camera(
                    glm::vec3(0.0f, -64.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            new en::Camera(
                    glm::vec3(0.0f, 0.0f, 64.0f),
                    glm::vec3(0.0f, 0.0f, -1.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            new en::Camera(
                    glm::vec3(0.0f, 0.0f, -64.0f),
                    glm::vec3(0.0f, 0.0f, 1.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f)
    };

    // Init rendering pipeline
    en::vk::Swapchain* swapchain = nullptr;
    if (en::Window::IsSupported())
    {
        swapchain = new en::vk::Swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);
    }

    nrcHpmRenderer = new en::NrcHpmRenderer(
            width,
            height,
            appConfig.trainSampleRatio,
            appConfig.trainSpp,
            false,
            &camera,
            hpmScene,
            nrc);

    mcHpmRenderer = new en::McHpmRenderer(width, height, 32, false, &camera, hpmScene);
    gtRenderer = new en::McHpmRenderer(width, height, 64, true, &camera, hpmScene);

    if (en::Window::IsSupported())
    {
        en::ImGuiRenderer::Init(width, height);
        switch (rendererId)
        {
            case 0: // MC
                en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
                break;
            case 1: // NRC
                en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
                break;
            default: // Error
                en::Log::Error("Renderer ID is invalid", true);
                break;
        }
    }

    // Swapchain rerecording because imgui renderer is now available
    if (en::Window::IsSupported()) { swapchain->Resize(width, height); }

    en::Log::Info(std::to_string(en::VulkanAPI::GetTimestampPeriod()));

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    bool shutdown = false;
    bool restartAfterClose = false;
    bool benchmark = true;
    bool continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
    while (continueLoop && !shutdown)
    {
        // Exit
        //if (frameCount == 10) { break; }

        // Update
        if (en::Window::IsSupported())
        {
            en::Window::Update();
            en::Input::Update();
        }
        en::Time::Update();

        if (en::Window::IsSupported())
        {
            width = en::Window::GetWidth();
            height = en::Window::GetHeight();
        }

        float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
        uint32_t fps = en::Time::GetFps();

        // Physics
        if (en::Window::IsSupported())
        {
            en::Input::HandleUserCamInput(&camera, deltaTime);
            camera.SetAspectRatio(width, height);
        }
        camera.UpdateUniformBuffer();

        // Render
        // Always render nrc for training
//        nrcHpmRenderer->Render(queue);
//        result = vkQueueWaitIdle(queue);
//        ASSERT_VULKAN(result);
//        nrcHpmRenderer->EvaluateTimestampQueries();

        switch (rendererId)
        {
            case 0: // MC
                mcHpmRenderer->Render(queue);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                mcHpmRenderer->EvaluateTimestampQueries();
                break;
            case 1: // NRC
                nrcHpmRenderer->Render(queue);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                nrcHpmRenderer->EvaluateTimestampQueries();
                break;
            default: // Error
                en::Log::Error("Renderer ID is invalid", true);
                break;
        }

        // Imgui
        if (en::Window::IsSupported())
        {
            en::ImGuiRenderer::StartFrame();

            ImGui::Begin("Statistics");
            ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
            ImGui::Text("DeltaTime %f", deltaTime);
            ImGui::Text("FPS %d", fps);
            ImGui::Text("NRC Loss %f", nrc.GetLoss());
            ImGui::End();

            ImGui::Begin("Controls");
            shutdown = ImGui::Button("Shutdown");
            ImGui::Checkbox("Restart after shutdown", &restartAfterClose);
            ImGui::Checkbox("Benchmark", &benchmark);

            if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
            {
                for (int i = 0; i < rendererMenuItems.size(); i++)
                {
                    bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
                    if (ImGui::Selectable(rendererMenuItems[i], selected))
                    {
                        if (i != rendererId)
                        {
                            rendererId = i;
                            switch (rendererId)
                            {
                                case 0: // MC
                                    en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
                                    break;
                                case 1: // NRC
                                    en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
                                    break;
                                default: // Error
                                    en::Log::Error("Renderer ID is invalid", true);
                                    break;
                            }
                        }
                        currentRendererMenuItem = rendererMenuItems[i];
                    };
                    if (selected) { ImGui::SetItemDefaultFocus(); }
                }
                ImGui::EndCombo();
            }

            ImGui::End();

            mcHpmRenderer->RenderImGui();
            nrcHpmRenderer->RenderImGui();

            hpmScene.Update(true);

            appConfig.RenderImGui();

            en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
            result = vkQueueWaitIdle(queue);
            ASSERT_VULKAN(result);
        }

        // Display
        if (en::Window::IsSupported()) { swapchain->DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE); }

        // Check loss
        if (benchmark && frameCount % 100 == 0)
        {
            en::Log::Info("Frame: " + std::to_string(frameCount));
            Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, &camera, queue);
        }

        //
        frameCount++;
        continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
    }

    // Evaluate at end
//    Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, queue);
//    std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
//    if (!std::filesystem::is_directory(outputDirPath) || std::filesystem::exists(outputDirPath))
//    {
//        std::filesystem::create_directory(outputDirPath);
//    }
//    std::string exrOutputFilePath =  outputDirPath + "1.exr";
//
//    // TODO: end evaluation
//    switch (rendererId)
//    {
//        case 0: // MC
//            mcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        case 1: // NRC
//            nrcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        default: // Error
//            en::Log::Error("Renderer ID is invalid", true);
//            break;
//    }

    // Stop gpu work
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    for (size_t i = 0; i < testCameras.size(); i++)
    {
        free(gtImages[i]);
    }

    gtRenderer->Destroy();
    delete gtRenderer;

    mcHpmRenderer->Destroy();
    delete mcHpmRenderer;

    nrcHpmRenderer->Destroy();
    delete nrcHpmRenderer;

    en::ImGuiRenderer::Shutdown();

    if (en::Window::IsSupported) { swapchain->Destroy(true); }

    for (size_t i = 0; i < testCameras.size(); i++)
    {
        testCameras[i]->Destroy();
        delete testCameras[i];
    }

    hpmScene.Destroy();
    camera.Destroy();
    nrc.Destroy();

    en::VulkanAPI::Shutdown();
    if (en::Window::IsSupported()) { en::Window::Shutdown(); }

    en::Log::Info("Ending " + appName);

    return restartAfterClose;
}

int main(int argc, char** argv)
{
    en::AppConfig appConfig(argc, argv);

    bool restartRunConfig;
    do {
        restartRunConfig = RunAppConfigInstance(appConfig);
    } while (restartRunConfig);

    return 0;
}
