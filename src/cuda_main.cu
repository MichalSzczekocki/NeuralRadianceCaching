#include "hip/hip_runtime.h"
#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <glm/glm.hpp>
#include <glm/gtc/random.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR;
PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR;

en::vk::CommandPool* commandPool;
VkCommandBuffer commandBuffer;

VkExternalMemoryHandleTypeFlagBits externalMemoryHandleType = VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;
size_t imageSizeInBytes;

VkImage image;
VkDeviceMemory imageMemory;
VkImageView imageView;
hipExternalMemory_t cuExtImageMemory;

VkExternalSemaphoreHandleTypeFlagBitsKHR externalSemaphoreHandleType = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;
VkSemaphore vkCudaStartSemaphore;
VkSemaphore vkCudaFinishedSemaphore;
hipExternalSemaphore_t cuCudaStartSemaphore;
hipExternalSemaphore_t cuCudaFinishedSemaphore;

hipStream_t streamToRun = 0;

HANDLE GetImageMemoryHandle(VkDevice device)
{
    HANDLE handle;

    VkMemoryGetWin32HandleInfoKHR vkMemoryGetWin32HandleInfoKHR = {};
    vkMemoryGetWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
    vkMemoryGetWin32HandleInfoKHR.pNext = NULL;
    vkMemoryGetWin32HandleInfoKHR.memory = imageMemory;
    vkMemoryGetWin32HandleInfoKHR.handleType = externalMemoryHandleType;

    fpGetMemoryWin32HandleKHR(device, &vkMemoryGetWin32HandleInfoKHR, &handle);
    return handle;
}

HANDLE GetSemaphoreHandle(VkDevice device, VkSemaphore vkSemaphore)
{
    HANDLE handle;

    VkSemaphoreGetWin32HandleInfoKHR vulkanSemaphoreGetWin32HandleInfoKHR = {};
    vulkanSemaphoreGetWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
    vulkanSemaphoreGetWin32HandleInfoKHR.pNext = NULL;
    vulkanSemaphoreGetWin32HandleInfoKHR.semaphore = vkSemaphore;
    vulkanSemaphoreGetWin32HandleInfoKHR.handleType = externalSemaphoreHandleType;

    fpGetSemaphoreWin32HandleKHR(device, &vulkanSemaphoreGetWin32HandleInfoKHR, &handle);
    return handle;
}

void LoadVulkanProcAddr()
{
    fpGetMemoryWin32HandleKHR = (PFN_vkGetMemoryWin32HandleKHR)vkGetInstanceProcAddr(
            en::VulkanAPI::GetInstance(),
            "vkGetMemoryWin32HandleKHR");

    fpGetSemaphoreWin32HandleKHR = (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(
            en::VulkanAPI::GetDevice(),
            "vkGetSemaphoreWin32HandleKHR");
}

void CreateCommandBuffer(uint32_t qfi)
{
    commandPool = new en::vk::CommandPool(VK_COMMAND_POOL_CREATE_RESET_COMMAND_BUFFER_BIT, qfi);
    commandPool->AllocateBuffers(1, VK_COMMAND_BUFFER_LEVEL_PRIMARY);
    commandBuffer = commandPool->GetBuffer(0);
}

void CreateImage(VkDevice device, VkQueue queue, uint32_t width, uint32_t height)
{
    VkFormat format = VK_FORMAT_R32G32B32A32_SFLOAT;
    imageSizeInBytes = width * height * 4 * sizeof(float);

    // Create Image
    VkExternalMemoryImageCreateInfo vkExternalMemImageCreateInfo = {};
    vkExternalMemImageCreateInfo.sType = VK_STRUCTURE_TYPE_EXTERNAL_MEMORY_IMAGE_CREATE_INFO;
    vkExternalMemImageCreateInfo.pNext = nullptr;
    vkExternalMemImageCreateInfo.handleTypes = externalMemoryHandleType;

    VkImageCreateInfo imageCI;
    imageCI.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
    imageCI.pNext = &vkExternalMemImageCreateInfo;
    imageCI.flags = 0;
    imageCI.imageType = VK_IMAGE_TYPE_2D;
    imageCI.format = format;
    imageCI.extent = { width, height, 1 };
    imageCI.mipLevels = 1;
    imageCI.arrayLayers = 1;
    imageCI.samples = VK_SAMPLE_COUNT_1_BIT;
    imageCI.tiling = VK_IMAGE_TILING_OPTIMAL;
    imageCI.usage = VK_IMAGE_USAGE_STORAGE_BIT | VK_IMAGE_USAGE_TRANSFER_SRC_BIT | VK_IMAGE_USAGE_TRANSFER_DST_BIT | VK_IMAGE_USAGE_SAMPLED_BIT;
    imageCI.sharingMode = VK_SHARING_MODE_EXCLUSIVE;
    imageCI.queueFamilyIndexCount = 0;
    imageCI.pQueueFamilyIndices = nullptr;
    imageCI.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;

    VkResult result = vkCreateImage(device, &imageCI, nullptr, &image);
    ASSERT_VULKAN(result);

    // Image Memory
    SECURITY_ATTRIBUTES winSecurityAttributes{};

    VkExportMemoryWin32HandleInfoKHR vulkanExportMemoryWin32HandleInfoKHR = {};
    vulkanExportMemoryWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_EXPORT_MEMORY_WIN32_HANDLE_INFO_KHR;
    vulkanExportMemoryWin32HandleInfoKHR.pNext = NULL;
    vulkanExportMemoryWin32HandleInfoKHR.pAttributes = &winSecurityAttributes;
    vulkanExportMemoryWin32HandleInfoKHR.dwAccess = DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportMemoryWin32HandleInfoKHR.name = (LPCWSTR)NULL;

    VkExportMemoryAllocateInfoKHR vulkanExportMemoryAllocateInfoKHR = {};
    vulkanExportMemoryAllocateInfoKHR.sType = VK_STRUCTURE_TYPE_EXPORT_MEMORY_ALLOCATE_INFO_KHR;
    vulkanExportMemoryAllocateInfoKHR.pNext = &vulkanExportMemoryWin32HandleInfoKHR;
    vulkanExportMemoryAllocateInfoKHR.handleTypes = externalMemoryHandleType;

    VkMemoryRequirements memoryRequirements;
    vkGetImageMemoryRequirements(device, image, &memoryRequirements);

    VkMemoryAllocateInfo allocateInfo;
    allocateInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
    allocateInfo.pNext = &vulkanExportMemoryAllocateInfoKHR;
    allocateInfo.allocationSize = memoryRequirements.size;
    allocateInfo.memoryTypeIndex = en::VulkanAPI::FindMemoryType(
            memoryRequirements.memoryTypeBits,
            VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT);

    result = vkAllocateMemory(device, &allocateInfo, nullptr, &imageMemory);
    ASSERT_VULKAN(result);

    result = vkBindImageMemory(device, image, imageMemory, 0);
    ASSERT_VULKAN(result);

    // Cuda image memory
    hipExternalMemoryHandleDesc cuExtMemHandleDesc{};
    cuExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    cuExtMemHandleDesc.handle.win32.handle = GetImageMemoryHandle(device);
    cuExtMemHandleDesc.size = imageSizeInBytes;

    hipError_t cudaResult = hipImportExternalMemory(&cuExtImageMemory, &cuExtMemHandleDesc);
    ASSERT_CUDA(cudaResult);

    // Change image layout
    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    ASSERT_VULKAN(result);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_GENERAL,
            VK_ACCESS_NONE,
            VK_ACCESS_SHADER_WRITE_BIT | VK_ACCESS_SHADER_READ_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_COMPUTE_SHADER_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    ASSERT_VULKAN(result);

    VkSubmitInfo submitInfo;
    submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
    submitInfo.pNext = nullptr;
    submitInfo.waitSemaphoreCount = 0;
    submitInfo.pWaitSemaphores = nullptr;
    submitInfo.pWaitDstStageMask = nullptr;
    submitInfo.commandBufferCount = 1;
    submitInfo.pCommandBuffers = &commandBuffer;
    submitInfo.signalSemaphoreCount = 0;
    submitInfo.pSignalSemaphores = nullptr;

    result = vkQueueSubmit(queue, 1, &submitInfo, VK_NULL_HANDLE);
    ASSERT_VULKAN(result);
    result = vkQueueWaitIdle(queue);
    ASSERT_VULKAN(result);

    // Create image view
    VkImageViewCreateInfo imageViewCI;
    imageViewCI.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
    imageViewCI.pNext = nullptr;
    imageViewCI.flags = 0;
    imageViewCI.image = image;
    imageViewCI.viewType = VK_IMAGE_VIEW_TYPE_2D;
    imageViewCI.format = format;
    imageViewCI.components.r = VK_COMPONENT_SWIZZLE_IDENTITY;
    imageViewCI.components.g = VK_COMPONENT_SWIZZLE_IDENTITY;
    imageViewCI.components.b = VK_COMPONENT_SWIZZLE_IDENTITY;
    imageViewCI.components.a = VK_COMPONENT_SWIZZLE_IDENTITY;
    imageViewCI.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
    imageViewCI.subresourceRange.baseArrayLayer = 0;
    imageViewCI.subresourceRange.baseMipLevel = 0;
    imageViewCI.subresourceRange.layerCount = 1;
    imageViewCI.subresourceRange.levelCount = 1;

    result = vkCreateImageView(device, &imageViewCI, nullptr, &imageView);
    ASSERT_VULKAN(result);
}

void CreateSyncObjects(VkDevice device)
{
    // Create vulkan semaphores
    SECURITY_ATTRIBUTES winSecurityAttributes;

    VkExportSemaphoreWin32HandleInfoKHR vulkanExportSemaphoreWin32HandleInfoKHR = {};
    vulkanExportSemaphoreWin32HandleInfoKHR.sType = VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_WIN32_HANDLE_INFO_KHR;
    vulkanExportSemaphoreWin32HandleInfoKHR.pNext = NULL;
    vulkanExportSemaphoreWin32HandleInfoKHR.pAttributes = &winSecurityAttributes;
    vulkanExportSemaphoreWin32HandleInfoKHR.dwAccess = DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportSemaphoreWin32HandleInfoKHR.name = (LPCWSTR)NULL;

    VkExportSemaphoreCreateInfoKHR vulkanExportSemaphoreCreateInfo = {};
    vulkanExportSemaphoreCreateInfo.sType = VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO_KHR;
    vulkanExportSemaphoreCreateInfo.pNext = &vulkanExportSemaphoreWin32HandleInfoKHR;
    vulkanExportSemaphoreCreateInfo.handleTypes = VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT;
    vulkanExportSemaphoreCreateInfo.pNext = NULL;
    vulkanExportSemaphoreCreateInfo.handleTypes = externalSemaphoreHandleType;

    VkSemaphoreCreateInfo semaphoreCI;
    semaphoreCI.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;
    semaphoreCI.pNext = &vulkanExportSemaphoreCreateInfo;
    semaphoreCI.flags = 0;

    VkResult result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &vkCudaStartSemaphore);
    ASSERT_VULKAN(result);

    result = vkCreateSemaphore(device, &semaphoreCI, nullptr, &vkCudaFinishedSemaphore);
    ASSERT_VULKAN(result);

    // Import vulkan semaphore to cuda
    hipExternalSemaphoreHandleDesc extCudaStartSemaphoreHD{};
    extCudaStartSemaphoreHD.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    extCudaStartSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, vkCudaStartSemaphore);

    hipError_t error = hipImportExternalSemaphore(&cuCudaStartSemaphore, &extCudaStartSemaphoreHD);
    ASSERT_CUDA(error);

    hipExternalSemaphoreHandleDesc extCudaFinishedSemaphoreHD{};
    extCudaFinishedSemaphoreHD.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    extCudaFinishedSemaphoreHD.handle.win32.handle = GetSemaphoreHandle(device, vkCudaFinishedSemaphore);

    error = hipImportExternalSemaphore(&cuCudaFinishedSemaphore, &extCudaFinishedSemaphoreHD);
    ASSERT_CUDA(error);
}

/*void InitDescriptor(VkDevice device)
{
	// Create desc set layout
	VkDescriptorSetLayoutBinding imageBinding;
	imageBinding.binding = 0;
	imageBinding.descriptorType = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
	imageBinding.descriptorCount = 1;
	imageBinding.stageFlags = VK_SHADER_STAGE_FRAGMENT_BIT;
	imageBinding.pImmutableSamplers = nullptr;

	VkDescriptorSetLayoutCreateInfo layoutCI;
	layoutCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
	layoutCI.pNext = nullptr;
	layoutCI.flags = 0;
	layoutCI.bindingCount = 1;
	layoutCI.pBindings = &imageBinding;

	VkResult result = vkCreateDescriptorSetLayout(device, &layoutCI, nullptr, &descSetLayout);
	ASSERT_VULKAN(result);

	// Create desc pool
	VkDescriptorPoolSize combinedImageSamplerPoolSize;
	combinedImageSamplerPoolSize.type = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
	combinedImageSamplerPoolSize.descriptorCount = 1;

	VkDescriptorPoolCreateInfo poolCI;
	poolCI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_POOL_CREATE_INFO;
	poolCI.pNext = nullptr;
	poolCI.flags = 0;
	poolCI.maxSets = 1;
	poolCI.poolSizeCount = 1;
	poolCI.pPoolSizes = &combinedImageSamplerPoolSize;

	result = vkCreateDescriptorPool(device, &poolCI, nullptr, &descPool);
	ASSERT_VULKAN(result);

	// Allocate desc set
	VkDescriptorSetAllocateInfo descSetAI;
	descSetAI.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
	descSetAI.pNext = nullptr;
	descSetAI.descriptorPool = descPool;
	descSetAI.descriptorSetCount = 1;
	descSetAI.pSetLayouts = &descSetLayout;

	result = vkAllocateDescriptorSets(device, &descSetAI, &descSet);
	ASSERT_VULKAN(result);

	// Update desc set
	VkDescriptorImageInfo imageInfo;
	imageInfo.sampler = sampler;
	imageInfo.imageView = imageView;
	imageInfo.imageLayout = VK_IMAGE_LAYOUT_GENERAL;

	VkWriteDescriptorSet imageWrite;
	imageWrite.sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
	imageWrite.pNext = nullptr;
	imageWrite.dstSet = descSet;
	imageWrite.dstBinding = 0;
	imageWrite.dstArrayElement = 0;
	imageWrite.descriptorCount = 1;
	imageWrite.descriptorType = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
	imageWrite.pImageInfo = &imageInfo;
	imageWrite.pBufferInfo = nullptr;
	imageWrite.pTexelBufferView = nullptr;

	vkUpdateDescriptorSets(device, 1, &imageWrite, 0, nullptr);
}*/

void DestroyVulkanResources(VkDevice device)
{
    vkDestroySemaphore(device, vkCudaFinishedSemaphore, nullptr);
    vkDestroySemaphore(device, vkCudaStartSemaphore, nullptr);

    vkDestroyImageView(device, imageView, nullptr);
    vkFreeMemory(device, imageMemory, nullptr);
    vkDestroyImage(device, image, nullptr);

    //vkDestroyDescriptorPool(device, descPool, nullptr);
    //vkDestroyDescriptorSetLayout(device, descSetLayout, nullptr);

    commandPool->Destroy();
    delete commandPool;
}

void CuVkSemaphoreWait(hipExternalSemaphore_t& extSemaphore)
{
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags = 0;

    hipError_t error = hipWaitExternalSemaphoresAsync(&extSemaphore, &extSemaphoreWaitParams, 1, streamToRun);
    ASSERT_CUDA(error);
}

void CuVkSemaphoreSignal(hipExternalSemaphore_t& extSemaphore)
{
    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags = 0;

    hipError_t error = hipSignalExternalSemaphoresAsync(&extSemaphore, &extSemaphoreSignalParams, 1, streamToRun);
    ASSERT_CUDA(error);
}

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");

    //uint32_t width = en::Window::GetWidth();
    //uint32_t height = en::Window::GetHeight();
    //nrcHpmRenderer->ResizeFrame(width, height);
    //en::ImGuiRenderer::Resize(width, height);
    //en::ImGuiRenderer::SetBackgroundImageView(imageView);
}

__global__ void CuFillImage(float* cuImageMemory)
{
    const int xIndex = threadIdx.x + (blockIdx.x * blockDim.x);
    const int yIndex = threadIdx.y + (blockIdx.y * blockDim.y);
    const int zIndex = threadIdx.z + (blockIdx.z * blockDim.z);
    const int pixelIndex = (yIndex * 768) + xIndex;
    const int pixelChannelIndex = (pixelIndex * 4) + zIndex;
    cuImageMemory[pixelChannelIndex] = 0.5f;
}

void RunTcnn()
{
    // Start engine
    const std::string appName("Neural-Radiance-Cache");
    uint32_t width = 768; // Multiple of 128 for nrc batch size
    uint32_t height = width;
    en::Log::Info("Starting " + appName);
    en::Window::Init(width, height, false, appName);
    en::VulkanAPI::Init(appName);
    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    en::vk::Swapchain swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);

    LoadVulkanProcAddr();
    CreateCommandBuffer(en::VulkanAPI::GetGraphicsQFI());
    CreateImage(device, queue, width, height);
    CreateSyncObjects(device);

    en::ImGuiRenderer::Init(width, height);
    en::ImGuiRenderer::SetBackgroundImageView(imageView);

    // Swapchain rerecording because imgui renderer is now available
    swapchain.Resize(width, height);

    /*// Init tcnn
	nlohmann::json config = {
	{"loss", {
		{"otype", "L2"}
	}},
	{"optimizer", {
		{"otype", "Adam"},
		{"learning_rate", 1e-3},
	}},
	{"encoding", {
        {"otype", "Composite"},
        {"reduction", "Concatenation"},
        {"nested", {
            {
                {"otype", "HashGrid"},
                {"n_dims_to_encode", 3},
                {"n_levels", 16},
                {"n_features_per_level", 2},
                {"log2_hashmap_size", 19},
                {"base_resolution", 16},
                {"per_level_scale", 2.0},
            },
            {
                {"otype", "OneBlob"},
                {"n_dims_to_encode", 2},
                {"n_bins", 4},
            },
        }},
	}},
	{"network", {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 128},
		{"n_hidden_layers", 6},
	}},
	};

    const uint32_t n_input_dims = 5;
    const uint32_t n_output_dims = 3;
    const uint32_t n_inference_steps = 36;
	const uint32_t n_training_steps = 10;
    const uint32_t batch_size = 16384;

    tcnn::TrainableModel model = tcnn::create_from_config(n_input_dims, n_output_dims, config);

    tcnn::GPUMatrix<float> training_batch_inputs(n_input_dims, batch_size);
    tcnn::GPUMatrix<float> training_batch_targets(n_output_dims, batch_size);

    tcnn::GPUMatrix<float> inference_inputs(n_input_dims, batch_size);
    tcnn::GPUMatrix<float> inference_outputs(n_output_dims, batch_size);

    tcnn::GPUMemory<uint8_t> tcnnMemory(batch_size * n_input_dims * sizeof(float));*/

    hipExternalMemoryBufferDesc cudaExtBufferDesc{};
    cudaExtBufferDesc.offset = 0;
    cudaExtBufferDesc.size = imageSizeInBytes;
    cudaExtBufferDesc.flags = 0;

    void* cuImageMemory;
    hipError_t error = hipExternalMemoryGetMappedBuffer(&cuImageMemory, cuExtImageMemory, &cudaExtBufferDesc);
    ASSERT_CUDA(error);

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    while (!en::Window::IsClosed())
    {
        // Update
        en::Window::Update();
        width = en::Window::GetWidth();
        height = en::Window::GetHeight();

        // Render frame
        if (frameCount > 0)
        {
            // Wait for vulkan
            const glm::vec4 randomColor = glm::linearRand(glm::vec4(0.0), glm::vec4(1.0));
            CuVkSemaphoreWait(cuCudaStartSemaphore);

            // Cuda rendering
            dim3 threads(2, 4, 4);
            dim3 blocks(width / 2, height / 4, 1);
            CuFillImage<<<blocks, threads, 0, streamToRun>>>(reinterpret_cast<float*>(cuImageMemory));
        }
        // Tell vulkan that cuda finished
        CuVkSemaphoreSignal(cuCudaFinishedSemaphore);

        // Imgui
        en::ImGuiRenderer::StartFrame();

        ImGui::Begin("Hello world");

        ImGui::End();

        // Display
        en::ImGuiRenderer::EndFrame(queue, vkCudaFinishedSemaphore);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        swapchain.DrawAndPresent(VK_NULL_HANDLE, vkCudaStartSemaphore);
        frameCount++;
    }
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    DestroyVulkanResources(device);

    en::ImGuiRenderer::Shutdown();

    swapchain.Destroy(true);

    en::VulkanAPI::Shutdown();
    en::Window::Shutdown();

    en::Log::Info("Ending " + appName);
}
