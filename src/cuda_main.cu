//#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <filesystem>
#include <engine/graphics/renderer/McHpmRenderer.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");
}

void Benchmark(
        uint32_t width,
        uint32_t height,
        uint32_t sceneID,
        const en::AppConfig& appConfig,
        const en::HpmScene& scene,
        VkQueue queue)
{
    // Create benchmark camera
    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

    std::array<en::Camera, 6> cameras = {
            en::Camera(
                    glm::vec3(64.0f, 0.0f, 0.0f),
                    glm::vec3(-1.0f, 0.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            en::Camera(
                    glm::vec3(-64.0f, 0.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            en::Camera(
                    glm::vec3(0.0f, 64.0f, 0.0f),
                    glm::vec3(0.0f, -1.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            en::Camera(
                    glm::vec3(0.0f, -64.0f, 0.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    glm::vec3(1.0f, 0.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            en::Camera(
                    glm::vec3(0.0f, 0.0f, 64.0f),
                    glm::vec3(0.0f, 0.0f, -1.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
            en::Camera(
                    glm::vec3(0.0f, 0.0f, -64.0f),
                    glm::vec3(0.0f, 0.0f, 1.0f),
                    glm::vec3(0.0f, 1.0f, 0.0f),
                    aspectRatio,
                    glm::radians(60.0f),
                    0.1f,
                    100.0f),
    };

    // Create output path if not exists
    std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
    if (!std::filesystem::is_directory(outputDirPath) || !std::filesystem::exists(outputDirPath))
    {
        std::filesystem::create_directory(outputDirPath);
    }

    // Create reference folder if not exists
    std::string referenceDirPath = "output/" + std::to_string(sceneID) + "/";
    if (!std::filesystem::is_directory(referenceDirPath) || !std::filesystem::exists(referenceDirPath))
    {
        en::Log::Info("Reference folder for scene " + std::to_string(sceneID) + " was not found. Creating reference images");

        // Create folder
        std::filesystem::create_directory(referenceDirPath);

        // Create ground truth renderer
        en::McHpmRenderer* gtRenderer = nullptr;

        for (size_t i = 0; i < cameras.size(); i++)
        {
            en::Log::Info("Generating reference image " + std::to_string(i));

            // Set new camera
            if (gtRenderer == nullptr) { gtRenderer = new en::McHpmRenderer(width, height, 64, &cameras[i], scene); }
            else { gtRenderer->SetCamera(&cameras[i]); }

            // Generate reference image
            for (size_t frame = 0; frame < 100; frame++)
            {
                gtRenderer->Render(queue);
                ASSERT_VULKAN(vkQueueWaitIdle(queue));
            }

            // Export reference image
            gtRenderer->ExportOutputImageToFile(queue, referenceDirPath + std::to_string(i) + ".exr");
        }
        // Destroy resources
        gtRenderer->Destroy();
        delete gtRenderer;
    }

    nrcHpmRenderer->Render(queue);
    ASSERT_VULKAN(vkQueueWaitIdle(queue));
    nrcHpmRenderer->ExportOutputImageToFile(queue, outputDirPath + "nrc_1.exr");

    mcHpmRenderer->Render(queue);
    ASSERT_VULKAN(vkQueueWaitIdle(queue));
    mcHpmRenderer->ExportOutputImageToFile(queue, outputDirPath + "mc_1.exr");

    // Destroy resources
    for (size_t i = 0; i < cameras.size(); i++) { cameras[i].Destroy(); }
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
    // Start engine
    const std::string appName("NeuralRadianceCaching");
    uint32_t width = appConfig.renderWidth;
    uint32_t height = appConfig.renderHeight;
    en::Log::Info("Starting " + appName);
    en::Window::Init(width, height, false, appName);
    en::Input::Init(en::Window::GetGLFWHandle());
    en::VulkanAPI::Init(appName);

    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    // Renderer select
    const std::vector<char*> rendererMenuItems = { "MC", "NRC" }; // TODO: Restir
    const char* currentRendererMenuItem = rendererMenuItems[1];
    uint32_t rendererId = 1;

    // Init resources
    en::NeuralRadianceCache nrc(appConfig);

    en::HpmScene hpmScene(appConfig);

    // Setup rendering
    en::Camera camera(
            glm::vec3(64.0f, 0.0f, 0.0f),
            glm::vec3(-1.0f, 0.0f, 0.0f),
            glm::vec3(0.0f, 1.0f, 0.0f),
            static_cast<float>(width) / static_cast<float>(height),
            glm::radians(60.0f),
            0.1f,
            100.0f);

    // Init rendering pipeline
    en::vk::Swapchain swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);

    nrcHpmRenderer = new en::NrcHpmRenderer(
            width,
            height,
            0.05f,
            1,
            &camera,
            hpmScene,
            nrc);

    mcHpmRenderer = new en::McHpmRenderer(width, height, 32, &camera, hpmScene);

    en::ImGuiRenderer::Init(width, height);
    switch (rendererId)
    {
        case 0: // MC
            en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
            break;
        case 1: // NRC
            en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
            break;
        default: // Error
            en::Log::Error("Renderer ID is invalid", true);
            break;
    }

//    en::ImGuiRenderer::SetBackgroundImageView(
//#ifdef NRC
//            nrcHpmRenderer->GetImageView()
//#else
//            mcHpmRenderer->GetImageView()
//#endif
//    );

    // Swapchain rerecording because imgui renderer is now available
    swapchain.Resize(width, height);

    en::Log::Info(std::to_string(en::VulkanAPI::GetTimestampPeriod()));

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    bool shutdown = false;
    bool restartAfterClose = false;
    while (!en::Window::IsClosed() && !shutdown)
    {
        // Exit
        if (frameCount == 1000) { break; }

        // Update
        en::Window::Update();
        en::Input::Update();
        en::Time::Update();

        width = en::Window::GetWidth();
        height = en::Window::GetHeight();

        float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
        uint32_t fps = en::Time::GetFps();
        en::Window::SetTitle(appName + " | Delta time: " + std::to_string(deltaTime) + "s | Fps: " + std::to_string(fps));

        if (frameCount % 100 == 0)
        {
            en::Log::Info("Loss: " + std::to_string(nrc.GetLoss()));
        }

        // Physics
        en::Input::HandleUserCamInput(&camera, deltaTime);
        camera.SetAspectRatio(width, height);
        camera.UpdateUniformBuffer();

        // Render
        // Always render nrc for training
//        nrcHpmRenderer->Render(queue);
//        result = vkQueueWaitIdle(queue);
//        ASSERT_VULKAN(result);
//        nrcHpmRenderer->EvaluateTimestampQueries();

        switch (rendererId)
        {
            case 0: // MC
                mcHpmRenderer->Render(queue);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                mcHpmRenderer->EvaluateTimestampQueries();
                break;
            case 1: // NRC
                nrcHpmRenderer->Render(queue);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                nrcHpmRenderer->EvaluateTimestampQueries();
                break;
            default: // Error
                en::Log::Error("Renderer ID is invalid", true);
                break;
        }

//#ifdef NRC
//        nrcHpmRenderer->Render(queue);
//        result = vkQueueWaitIdle(queue);
//        ASSERT_VULKAN(result);
//        nrcHpmRenderer->EvaluateTimestampQueries();
//#else
//        mcHpmRenderer->Render(queue);
//		result = vkQueueWaitIdle(queue);
//		ASSERT_VULKAN(result);
//		mcHpmRenderer->EvaluateTimestampQueries();
//#endif

        // Imgui
        en::ImGuiRenderer::StartFrame();

        ImGui::Begin("Statistics");
        ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
        ImGui::Text("DeltaTime %f", deltaTime);
        ImGui::Text("FPS %d", fps);
        ImGui::Text("NRC Loss %f", nrc.GetLoss());
        ImGui::End();

        ImGui::Begin("Controls");
        shutdown = ImGui::Button("Shutdown");
        ImGui::Checkbox("Restart after shutdown", &restartAfterClose);

        if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
        {
            for (int i = 0; i < rendererMenuItems.size(); i++)
            {
                bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
                if (ImGui::Selectable(rendererMenuItems[i], selected))
                {
                    if (i != rendererId)
                    {
                        rendererId = i;
                        switch (rendererId)
                        {
                            case 0: // MC
                                en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
                                break;
                            case 1: // NRC
                                en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
                                break;
                            default: // Error
                                en::Log::Error("Renderer ID is invalid", true);
                                break;
                        }
                    }
                    currentRendererMenuItem = rendererMenuItems[i];
                };
                if (selected) { ImGui::SetItemDefaultFocus(); }
            }
            ImGui::EndCombo();
        }

        ImGui::End();

        mcHpmRenderer->RenderImGui();
        nrcHpmRenderer->RenderImGui();

//        switch (rendererId)
//        {
//            case 0: // MC
//                break;
//            case 1: // NRC
//                nrcHpmRenderer->RenderImGui();
//                break;
//            default: // Error
//                en::Log::Error("Renderer ID is invalid", true);
//                break;
//        }

//#ifdef NRC
//        nrcHpmRenderer->RenderImGui();
//#endif

        hpmScene.Update(true);

        appConfig.RenderImGui();

        en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
        result = vkQueueWaitIdle(queue);
        ASSERT_VULKAN(result);

        // Display
        swapchain.DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE);
        frameCount++;
    }

    // Evaluate at end
    Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, queue);
//    std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
//    if (!std::filesystem::is_directory(outputDirPath) || std::filesystem::exists(outputDirPath))
//    {
//        std::filesystem::create_directory(outputDirPath);
//    }
//    std::string exrOutputFilePath =  outputDirPath + "1.exr";
//
//    // TODO: end evaluation
//    switch (rendererId)
//    {
//        case 0: // MC
//            mcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        case 1: // NRC
//            nrcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        default: // Error
//            en::Log::Error("Renderer ID is invalid", true);
//            break;
//    }

    // Stop gpu work
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    mcHpmRenderer->Destroy();
    delete mcHpmRenderer;

    nrcHpmRenderer->Destroy();
    delete nrcHpmRenderer;

    en::ImGuiRenderer::Shutdown();

    swapchain.Destroy(true);

    hpmScene.Destroy();
    camera.Destroy();
    nrc.Destroy();

    en::VulkanAPI::Shutdown();
    en::Window::Shutdown();

    en::Log::Info("Ending " + appName);

    return restartAfterClose;
}

int main(int argc, char** argv)
{
    en::AppConfig appConfig(argc, argv);

    bool restartRunConfig;
    do {
        restartRunConfig = RunAppConfigInstance(appConfig);
    } while (restartRunConfig);

    return 0;
}
