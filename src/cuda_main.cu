//#include <cuda_main.hpp>

#define VK_USE_PLATFORM_WIN32_KHR

#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>

#include <engine/util/Log.hpp>
#include <engine/graphics/Window.hpp>
#include <engine/graphics/VulkanAPI.hpp>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/vulkan/CommandRecorder.hpp>
#include <engine/graphics/renderer/ImGuiRenderer.hpp>
#include <engine/graphics/vulkan/Swapchain.hpp>
#include <imgui.h>
#include <engine/graphics/renderer/NrcHpmRenderer.hpp>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <engine/util/read_file.hpp>
#include <engine/util/Input.hpp>
#include <engine/util/Time.hpp>
#include <engine/HpmScene.hpp>
#include <engine/AppConfig.hpp>
#include <filesystem>
#include <engine/graphics/renderer/McHpmRenderer.hpp>
#include <tinyexr.h>
#include <engine/graphics/vulkan/CommandPool.hpp>
#include <engine/graphics/Reference.hpp>

#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <vulkan/vulkan.h>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

en::Reference* reference = nullptr;
en::NrcHpmRenderer* nrcHpmRenderer = nullptr;
en::McHpmRenderer* mcHpmRenderer = nullptr;

void RecordSwapchainCommandBuffer(VkCommandBuffer commandBuffer, VkImage image)
{
    uint32_t width = en::Window::GetWidth();
    uint32_t height = en::Window::GetHeight();

    VkCommandBufferBeginInfo beginInfo;
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.pNext = nullptr;
    beginInfo.flags = 0;
    beginInfo.pInheritanceInfo = nullptr;

    VkResult result = vkBeginCommandBuffer(commandBuffer, &beginInfo);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to begin VkCommandBuffer", true);

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_UNDEFINED,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_ACCESS_NONE_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT);

    if (nrcHpmRenderer != nullptr && mcHpmRenderer != nullptr && en::ImGuiRenderer::IsInitialized())
    {
        VkImageCopy imageCopy;
        imageCopy.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.srcSubresource.mipLevel = 0;
        imageCopy.srcSubresource.baseArrayLayer = 0;
        imageCopy.srcSubresource.layerCount = 1;
        imageCopy.srcOffset = { 0, 0, 0 };
        imageCopy.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
        imageCopy.dstSubresource.mipLevel = 0;
        imageCopy.dstSubresource.baseArrayLayer = 0;
        imageCopy.dstSubresource.layerCount = 1;
        imageCopy.dstOffset = { 0, 0, 0 };
        imageCopy.extent = { width, height, 1 };

        vkCmdCopyImage(
                commandBuffer,
                en::ImGuiRenderer::GetImage(),
                VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                image,
                VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
                1,
                &imageCopy);
    }

    en::vk::CommandRecorder::ImageLayoutTransfer(
            commandBuffer,
            image,
            VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,
            VK_IMAGE_LAYOUT_PRESENT_SRC_KHR,
            VK_ACCESS_TRANSFER_WRITE_BIT,
            VK_ACCESS_COLOR_ATTACHMENT_READ_BIT,
            VK_PIPELINE_STAGE_TRANSFER_BIT,
            VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT);

    result = vkEndCommandBuffer(commandBuffer);
    if (result != VK_SUCCESS)
        en::Log::Error("Failed to end VkCommandBuffer", true);
}

void SwapchainResizeCallback()
{
    en::Window::WaitForUsableSize();
    vkDeviceWaitIdle(en::VulkanAPI::GetDevice()); // TODO: causes error with multithreaded rendering

    en::Log::Info("Skipping swapchain resize callback");
}

void Benchmark(const en::Camera* camera, VkQueue queue, size_t frameCount)
{
    en::Log::Info("Frame: " + std::to_string(frameCount));
    reference->CompareNrc(*nrcHpmRenderer, camera, queue);
    //reference->CompareMc(*mcHpmRenderer, camera, queue);
}

bool RunAppConfigInstance(const en::AppConfig& appConfig)
{
    // Start engine
    const std::string appName("NeuralRadianceCaching");
    uint32_t width = 1920;
    uint32_t height = 1080;
    en::Log::Info("Starting " + appName);

    en::Window::Init(width, height, false, appName);
    if (en::Window::IsSupported()) { en::Input::Init(en::Window::GetGLFWHandle()); }
    en::VulkanAPI::Init(appName);
    const VkDevice device = en::VulkanAPI::GetDevice();
    const uint32_t qfi = en::VulkanAPI::GetGraphicsQFI();
    const VkQueue queue = en::VulkanAPI::GetGraphicsQueue();

    // Renderer select
    const std::vector<char*> rendererMenuItems = { "MC", "NRC" }; // TODO: Restir
    const char* currentRendererMenuItem = rendererMenuItems[1];
    uint32_t rendererId = 1;

    // Init resources
    en::NeuralRadianceCache nrc(appConfig);

    en::HpmScene hpmScene(appConfig);

    // Setup rendering
    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);
    en::Camera camera(
            glm::vec3(64.0f, 0.0f, 0.0f),
            glm::vec3(-1.0f, 0.0f, 0.0f),
            glm::vec3(0.0f, 1.0f, 0.0f),
            aspectRatio,
            glm::radians(60.0f),
            0.1f,
            100.0f);

    // Init reference
    reference = new en::Reference(width, height, appConfig, hpmScene, queue);

    // Init rendering pipeline
    en::vk::Swapchain* swapchain = nullptr;
    if (en::Window::IsSupported())
    {
        swapchain = new en::vk::Swapchain(width, height, RecordSwapchainCommandBuffer, SwapchainResizeCallback);
    }

    nrcHpmRenderer = new en::NrcHpmRenderer(
            width,
            height,
            appConfig.trainSampleRatio,
            appConfig.trainSpp,
            appConfig.primaryRayLength,
            false,
            &camera,
            hpmScene,
            nrc);

    mcHpmRenderer = new en::McHpmRenderer(width, height, 32, false, &camera, hpmScene);

    if (en::Window::IsSupported())
    {
        en::ImGuiRenderer::Init(width, height);
        switch (rendererId)
        {
            case 0: // MC
                en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
                break;
            case 1: // NRC
                en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
                break;
            default: // Error
                en::Log::Error("Renderer ID is invalid", true);
                break;
        }
    }

    // Swapchain rerecording because imgui renderer is now available
    if (en::Window::IsSupported()) { swapchain->Resize(width, height); }

    en::Log::Info(std::to_string(en::VulkanAPI::GetTimestampPeriod()));

    // Main loop
    VkResult result;
    size_t frameCount = 0;
    bool shutdown = false;
    bool restartAfterClose = false;
    bool benchmark = true;
    bool continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
    while (continueLoop && !shutdown)
    {
        // Update
        if (en::Window::IsSupported())
        {
            en::Window::Update();
            en::Input::Update();
        }
        en::Time::Update();

        if (en::Window::IsSupported())
        {
            width = en::Window::GetWidth();
            height = en::Window::GetHeight();
        }

        float deltaTime = static_cast<float>(en::Time::GetDeltaTime());
        uint32_t fps = en::Time::GetFps();

        // Physics
        if (en::Window::IsSupported())
        {
            en::Input::HandleUserCamInput(&camera, deltaTime);
            camera.SetAspectRatio(width, height);
        }
        camera.UpdateUniformBuffer();

        // Render
        switch (rendererId)
        {
            case 0: // MC
                mcHpmRenderer->Render(queue);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                mcHpmRenderer->EvaluateTimestampQueries();
                break;
            case 1: // NRC
                nrcHpmRenderer->Render(queue, true);
                result = vkQueueWaitIdle(queue);
                ASSERT_VULKAN(result);
                nrcHpmRenderer->EvaluateTimestampQueries();
                break;
            default: // Error
                en::Log::Error("Renderer ID is invalid", true);
                break;
        }

        // Imgui
        if (en::Window::IsSupported())
        {
            en::ImGuiRenderer::StartFrame();

            ImGui::Begin("Statistics");
            ImGui::Text((std::string("Framecount ") + std::to_string(frameCount)).c_str());
            ImGui::Text("DeltaTime %f", deltaTime);
            ImGui::Text("FPS %d", fps);
            ImGui::Text("NRC Loss %f", nrc.GetLoss());
            ImGui::End();

            ImGui::Begin("Controls");
            shutdown = ImGui::Button("Shutdown");
            ImGui::Checkbox("Restart after shutdown", &restartAfterClose);
            ImGui::Checkbox("Benchmark", &benchmark);

            if (ImGui::BeginCombo("##combo", currentRendererMenuItem))
            {
                for (int i = 0; i < rendererMenuItems.size(); i++)
                {
                    bool selected = (currentRendererMenuItem == rendererMenuItems[i]);
                    if (ImGui::Selectable(rendererMenuItems[i], selected))
                    {
                        if (i != rendererId)
                        {
                            rendererId = i;
                            switch (rendererId)
                            {
                                case 0: // MC
                                    en::ImGuiRenderer::SetBackgroundImageView(mcHpmRenderer->GetImageView());
                                    break;
                                case 1: // NRC
                                    en::ImGuiRenderer::SetBackgroundImageView(nrcHpmRenderer->GetImageView());
                                    break;
                                default: // Error
                                    en::Log::Error("Renderer ID is invalid", true);
                                    break;
                            }
                        }
                        currentRendererMenuItem = rendererMenuItems[i];
                    };
                    if (selected) { ImGui::SetItemDefaultFocus(); }
                }
                ImGui::EndCombo();
            }

            ImGui::End();

            mcHpmRenderer->RenderImGui();
            nrcHpmRenderer->RenderImGui();

            hpmScene.Update(true);

            appConfig.RenderImGui();

            en::ImGuiRenderer::EndFrame(queue, VK_NULL_HANDLE);
            result = vkQueueWaitIdle(queue);
            ASSERT_VULKAN(result);
        }

        // Display
        if (en::Window::IsSupported()) { swapchain->DrawAndPresent(VK_NULL_HANDLE, VK_NULL_HANDLE); }

        // Check loss
//        if (benchmark && frameCount % 100 == 0)
//        {
//            en::Log::Info("Frame: " + std::to_string(frameCount));
//            Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, &camera, queue);
//        }
        // Benchmark
        if (benchmark && frameCount % 10 == 0) { Benchmark(&camera, queue, frameCount); }

        //
        frameCount++;
        continueLoop = en::Window::IsSupported() ? !en::Window::IsClosed() : true;
    }

    // Evaluate at end
//    Benchmark(appConfig.renderWidth, appConfig.renderHeight, appConfig.scene.id, appConfig, hpmScene, queue);
//    std::string outputDirPath = "output/ " + appConfig.GetName() + "/";
//    if (!std::filesystem::is_directory(outputDirPath) || std::filesystem::exists(outputDirPath))
//    {
//        std::filesystem::create_directory(outputDirPath);
//    }
//    std::string exrOutputFilePath =  outputDirPath + "1.exr";
//
//    // TODO: end evaluation
//    switch (rendererId)
//    {
//        case 0: // MC
//            mcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        case 1: // NRC
//            nrcHpmRenderer->ExportImageToFile(queue, exrOutputFilePath);
//            break;
//        default: // Error
//            en::Log::Error("Renderer ID is invalid", true);
//            break;
//    }

    // Stop gpu work
    result = vkDeviceWaitIdle(device);
    ASSERT_VULKAN(result);

    // End
    mcHpmRenderer->Destroy();
    delete mcHpmRenderer;

    nrcHpmRenderer->Destroy();
    delete nrcHpmRenderer;

    en::ImGuiRenderer::Shutdown();

    if (en::Window::IsSupported) { swapchain->Destroy(true); }

    reference->Destroy();
    delete reference;

    hpmScene.Destroy();
    camera.Destroy();
    nrc.Destroy();

    en::VulkanAPI::Shutdown();
    if (en::Window::IsSupported()) { en::Window::Shutdown(); }

    en::Log::Info("Ending " + appName);

    return restartAfterClose;
}

int main(int argc, char** argv)
{
    std::vector<char*> myargv(argc);
    std::memcpy(myargv.data(), argv, sizeof(char*) * argc);
    myargv = {
            "NeuralRadianceCaching",
            "RelativeL2", "Adam", "0.001", "0.99",
            "0", "0",
            "64", "4", "15",
            "0",
            "0.05", "1", "3"};
    en::AppConfig appConfig(myargv);

    bool restartRunConfig;
    do {
        restartRunConfig = RunAppConfigInstance(appConfig);
    } while (restartRunConfig);

    return 0;
}
