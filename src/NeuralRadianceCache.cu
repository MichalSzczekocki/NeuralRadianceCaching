#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <random>
#include <engine/util/Log.hpp>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

namespace en
{
    NeuralRadianceCache::NeuralRadianceCache(
            const nlohmann::json& config,
            uint32_t inputCount,
            uint32_t outputCount,
            uint32_t log2BatchSize)
            :
            m_Model(tcnn::create_from_config(inputCount, outputCount, config)),
            m_InputCount(inputCount),
            m_OutputCount(outputCount),
            m_BatchSize(2 << (log2BatchSize - 1))
    {
    }

    void NeuralRadianceCache::Init(
            uint32_t inferCount,
            uint32_t trainCount,
            float* dCuInferInput,
            float* dCuInferOutput,
            float* dCuTrainInput,
            float* dCuTrainTarget,
            hipExternalSemaphore_t cudaStartSemaphore,
            hipExternalSemaphore_t cudaFinishedSemaphore)
    {
        // Init members
        m_CudaStartSemaphore = cudaStartSemaphore;
        m_CudaFinishedSemaphore = cudaFinishedSemaphore;

        // Check batch size compatibility
        if (inferCount % m_BatchSize != 0 || trainCount % m_BatchSize != 0)
        {
            Log::Error("NRC batch size is not compatible with infer count or train count", true);
        }

        // Init infer buffers
        uint32_t inferBatchCount = inferCount / m_BatchSize;
        m_InferInputBatches.resize(inferBatchCount);
        m_InferOutputBatches.resize(inferBatchCount);
        for (uint32_t i = 0; i < inferBatchCount; i++)
        {
            const size_t floatInputOffset = i * m_BatchSize * m_InputCount;
            const size_t floatOutputOffset = i * m_BatchSize * m_OutputCount;
            m_InferInputBatches[i] = tcnn::GPUMatrix<float>(&(dCuInferInput[floatInputOffset]), m_InputCount, m_BatchSize);
            m_InferOutputBatches[i] = tcnn::GPUMatrix<float>(&(dCuInferOutput[floatOutputOffset]), m_OutputCount, m_BatchSize);
        }

        // Init train buffers
        uint32_t trainBatchCount = trainCount / m_BatchSize;
        m_TrainInputBatches.resize(trainBatchCount);
        m_TrainTargetBatches.resize(trainBatchCount);
        for (uint32_t i = 0; i < trainBatchCount; i++)
        {
            const size_t floatInputOffset = i * m_BatchSize * m_InputCount;
            const size_t floatTargetOffset = i * m_BatchSize * m_OutputCount;
            m_TrainInputBatches[i] = tcnn::GPUMatrix<float>(&(dCuTrainInput[floatInputOffset]), m_InputCount, m_BatchSize);
            m_TrainTargetBatches[i] = tcnn::GPUMatrix<float>(&(dCuTrainTarget[floatTargetOffset]), m_OutputCount, m_BatchSize);
        }
    }

    void NeuralRadianceCache::InferAndTrain()
    {
        AwaitCudaStartSemaphore();
        Inference();
        Train();
        SignalCudaFinishedSemaphore();
    }

    void NeuralRadianceCache::Destroy()
    {
    }

    float NeuralRadianceCache::GetLoss() const
    {
        return m_Loss;
    }

    void NeuralRadianceCache::Inference()
    {
        for (size_t i = 0; i < m_InferInputBatches.size(); i++)
        {
            const tcnn::GPUMatrix<float>& inputBatch = m_InferInputBatches[i];
            tcnn::GPUMatrix<float>& outputBatch = m_InferOutputBatches[i];
            m_Model.network->inference(inputBatch, outputBatch);
        }
    }

    void NeuralRadianceCache::Train()
    {
        for (size_t i = 0; i < m_TrainInputBatches.size(); i++)
        {
            const tcnn::GPUMatrix<float>& inputBatch = m_TrainInputBatches[i];
            const tcnn::GPUMatrix<float>& targetBatch = m_TrainTargetBatches[i];
            auto forwardContext = m_Model.trainer->training_step(inputBatch, targetBatch);
            m_Loss = m_Model.trainer->loss(*forwardContext.get());
        }
    }

    void NeuralRadianceCache::AwaitCudaStartSemaphore()
    {
        hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
        memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
        extSemaphoreWaitParams.params.fence.value = 0;
        extSemaphoreWaitParams.flags = 0;

        hipError_t error = hipWaitExternalSemaphoresAsync(&m_CudaStartSemaphore, &extSemaphoreWaitParams, 1);
        ASSERT_CUDA(error);
    }

    void NeuralRadianceCache::SignalCudaFinishedSemaphore()
    {
        hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
        memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
        extSemaphoreSignalParams.params.fence.value = 0;
        extSemaphoreSignalParams.flags = 0;

        hipError_t error = hipSignalExternalSemaphoresAsync(&m_CudaFinishedSemaphore, &extSemaphoreSignalParams, 1);
        ASSERT_CUDA(error);
    }
}
