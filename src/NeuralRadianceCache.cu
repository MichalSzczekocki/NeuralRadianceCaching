#include <hip/hip_runtime.h>
#include <tiny-cuda-nn/config.h>
#include <engine/graphics/NeuralRadianceCache.hpp>
#include <random>
#include <engine/util/Log.hpp>

#define ASSERT_CUDA(error) if (error != hipSuccess) { en::Log::Error("Cuda assert triggered: " + std::string(hipGetErrorName(error)), true); }

namespace en
{
    const uint32_t NeuralRadianceCache::sc_InputCount = 5;
    const uint32_t NeuralRadianceCache::sc_OutputCount = 3;

    NeuralRadianceCache::NeuralRadianceCache(const AppConfig& appConfig) :
            m_BatchSize(2 << (appConfig.log2BatchSize - 1))
    {
        nlohmann::json modelConfig = {
                {"loss", {
                        {"otype", appConfig.lossFn}
                }},
                {"optimizer", {
                        {"otype", appConfig.optimizer},
                        {"learning_rate", appConfig.learningRate},
                }},
                appConfig.encoding.jsonConfig,
                {"network", {
                        {"otype", "FullyFusedMLP"},
                        {"activation", "ReLU"},
                        {"output_activation", "None"},
                        {"n_neurons", appConfig.nnWidth},
                        {"n_hidden_layers", appConfig.nnDepth},
                }},
        };

        m_Model = tcnn::create_from_config(sc_InputCount, sc_OutputCount, modelConfig);
    }

    void NeuralRadianceCache::Init(
            uint32_t inferCount,
            uint32_t trainCount,
            float* dCuInferInput,
            float* dCuInferOutput,
            float* dCuTrainInput,
            float* dCuTrainTarget,
            hipExternalSemaphore_t cudaStartSemaphore,
            hipExternalSemaphore_t cudaFinishedSemaphore)
    {
        // Check if sample counts are compatible
        if (inferCount % 16 != 0) { en::Log::Error("NRC requires inferCount to be a multiple of 16", true); }
        if (trainCount % 16 != 0) { en::Log::Error("NRC required trainCount to be a multiple of 16", true); }

        // Init members
        m_CudaStartSemaphore = cudaStartSemaphore;
        m_CudaFinishedSemaphore = cudaFinishedSemaphore;

        // Init infer buffers
        uint32_t inferBatchCount = inferCount / m_BatchSize;
        uint32_t inferLastBatchSize = inferCount - (inferBatchCount * m_BatchSize);
        m_InferInputBatches.resize(inferBatchCount);
        m_InferOutputBatches.resize(inferBatchCount);

        size_t floatInputOffset = 0;
        size_t floatOutputOffset = 0;
        for (uint32_t i = 0; i < inferBatchCount; i++)
        {
            m_InferInputBatches[i] = tcnn::GPUMatrix<float>(&(dCuInferInput[floatInputOffset]), sc_InputCount, m_BatchSize);
            m_InferOutputBatches[i] = tcnn::GPUMatrix<float>(&(dCuInferOutput[floatOutputOffset]), sc_OutputCount, m_BatchSize);
            floatInputOffset += m_BatchSize * sc_InputCount;
            floatOutputOffset += m_BatchSize * sc_OutputCount;
        }

        if (inferLastBatchSize > 0)
        {
            m_InferInputBatches.push_back(tcnn::GPUMatrix<float>(&(dCuInferInput[floatInputOffset]), sc_InputCount, inferLastBatchSize));
            m_InferOutputBatches.push_back(tcnn::GPUMatrix<float>(&(dCuInferOutput[floatOutputOffset]), sc_OutputCount, inferLastBatchSize));
        }

        // Init train buffers
        uint32_t trainBatchCount = trainCount / m_BatchSize;
        uint32_t trainLastBatchSize = trainCount - (trainBatchCount * m_BatchSize);
        m_TrainInputBatches.resize(trainBatchCount);
        m_TrainTargetBatches.resize(trainBatchCount);

        floatInputOffset = 0;
        floatOutputOffset = 0;
        for (uint32_t i = 0; i < trainBatchCount; i++)
        {
            m_TrainInputBatches[i] = tcnn::GPUMatrix<float>(&(dCuTrainInput[floatInputOffset]), sc_InputCount, m_BatchSize);
            m_TrainTargetBatches[i] = tcnn::GPUMatrix<float>(&(dCuTrainTarget[floatOutputOffset]), sc_OutputCount, m_BatchSize);
            floatInputOffset += m_BatchSize * sc_InputCount;
            floatOutputOffset += m_BatchSize * sc_OutputCount;
        }

        if (trainLastBatchSize > 0)
        {
            m_TrainInputBatches.push_back(tcnn::GPUMatrix<float>(&(dCuTrainInput[floatInputOffset]), sc_InputCount, trainLastBatchSize));
            m_TrainTargetBatches.push_back(tcnn::GPUMatrix<float>(&(dCuTrainTarget[floatOutputOffset]), sc_OutputCount, trainLastBatchSize));
        }
    }

    void NeuralRadianceCache::InferAndTrain(const uint32_t* inferFilter, bool train)
    {
        AwaitCudaStartSemaphore();
        Inference(inferFilter);
        if (train) { Train(); }
        SignalCudaFinishedSemaphore();
    }

    void NeuralRadianceCache::Destroy()
    {
    }

    float NeuralRadianceCache::GetLoss() const
    {
        return m_Loss;
    }

    size_t NeuralRadianceCache::GetInferBatchCount() const
    {
        return m_InferInputBatches.size();
    }

    size_t NeuralRadianceCache::GetTrainBatchCount() const
    {
        return m_TrainInputBatches.size();
    }

    uint32_t NeuralRadianceCache::GetBatchSize() const
    {
        return m_BatchSize;
    }

    void NeuralRadianceCache::Inference(const uint32_t* inferFilter)
    {
        for (size_t i = 0; i < m_InferInputBatches.size(); i++)
        {
            if (inferFilter[i] > 0)
            {
                const tcnn::GPUMatrix<float>& inputBatch = m_InferInputBatches[i];
                tcnn::GPUMatrix<float>& outputBatch = m_InferOutputBatches[i];
                m_Model.network->inference(inputBatch, outputBatch);
            }
        }
    }

    void NeuralRadianceCache::Train()
    {
        for (size_t i = 0; i < m_TrainInputBatches.size(); i++)
        {
            const tcnn::GPUMatrix<float>& inputBatch = m_TrainInputBatches[i];
            const tcnn::GPUMatrix<float>& targetBatch = m_TrainTargetBatches[i];
            auto forwardContext = m_Model.trainer->training_step(inputBatch, targetBatch);
            m_Loss = m_Model.trainer->loss(*forwardContext.get());
        }
    }

    void NeuralRadianceCache::AwaitCudaStartSemaphore()
    {
        hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
        memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
        extSemaphoreWaitParams.params.fence.value = 0;
        extSemaphoreWaitParams.flags = 0;

        hipError_t error = hipWaitExternalSemaphoresAsync(&m_CudaStartSemaphore, &extSemaphoreWaitParams, 1);
        ASSERT_CUDA(error);
    }

    void NeuralRadianceCache::SignalCudaFinishedSemaphore()
    {
        hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
        memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
        extSemaphoreSignalParams.params.fence.value = 0;
        extSemaphoreSignalParams.flags = 0;

        hipError_t error = hipSignalExternalSemaphoresAsync(&m_CudaFinishedSemaphore, &extSemaphoreSignalParams, 1);
        ASSERT_CUDA(error);
    }
}
